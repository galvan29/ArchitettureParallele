#include <stdio.h>
#include <iostream>
#include <cmath>
#include <fstream>
#include <bits/stdc++.h>
#include <numeric>
#include <hip/hip_runtime.h>
#include <omp.h>
#include <malloc.h>
#include <unistd.h>
#include <list>

using namespace std;

void funcRead(string str[])
{
  string myText;
  int p = 0;
  ifstream MyReadFile("vincoli.txt");
  while (getline(MyReadFile, myText))
  {
    str[p] = myText;
    p++;
  }
  MyReadFile.close();
}

string firstLine()
{
  ifstream infile("vincoli.txt");
  string sLine;
  if (infile.good())
  {
    getline(infile, sLine);
  }
  return sLine;
}

__device__ void diagonale(bool *d_matrix, int length, int thid)
{
  int secondo = (thid % length);
  int primo = floorf(thid / length);
  d_matrix[(primo * length) + secondo] |= d_matrix[(secondo * length) + primo];
}

__global__ void prova(bool *d_matrix, int length, long int lengthx2, int *d_posizione)
{
  int thid2 = blockIdx.x * blockDim.x + threadIdx.x;
  int thid = 0;
  /*if(thid2 == 0){
    printf("Ma quante volte rientro\n");
  }*/

  for (int Pass = 0; Pass < ceilf((lengthx2 / (blockDim.x * gridDim.x))) + 1; Pass++)
  {
    thid = thid2 + Pass * (gridDim.x * blockDim.x);

    if (thid < (lengthx2))
    {
      if (d_matrix[thid])
      { // thid = 4
        int secondo = (thid % length);
        if (secondo >= (length / 2))
          secondo = secondo - (length / 2);
        else
          secondo = secondo + (length / 2);                                 // 4
        int primo = floorf(thid / length); // 0
        for (int i = (secondo * length); i < ((secondo + 1) * length); i++)
        { // da 24
          if (d_matrix[i] && ((i % length) + 1) != (primo + 1))
          { // 24 però 24%6+1 == 1 quindi non entro
            int posizione = (primo * length) + (i % length);
            d_matrix[posizione] = 1;
            //d_posizione[0] = 1;
            atomicAdd(&d_posizione[0], 1.0f); 
          } 
        }
      }
    }
    __syncthreads();

    if (thid < (length * length))
    {
      diagonale(d_matrix, length, thid);
    }
  }
  __syncthreads();
}

// LA DIAGONALE NON MI TORNA
__global__ void checkDiagonale(bool *matrix, int length)
{
  int thid = blockIdx.x * blockDim.x + threadIdx.x;
  int thidCheck1 = 0;
  int thidCheck2 = 0;
  if (thid % (length + 1) == 0 && thid < (length * length))
  {
    thidCheck1 = thid;
    thidCheck2 = (length + 1) * (length / 2) + thid;
    if (matrix[thidCheck1] == 1 && matrix[thidCheck1] == matrix[thidCheck2])
    {
      printf("Nella posizione %d e nella posizione %d hanno entrambi 1\n", thidCheck1, thidCheck2);
      printf("Quindi teoricamente non ci sono soluzioni\n\n");
    }
  }
  if (thid == 0)
    printf("Ho controllato la diagonale\n");
  __syncthreads();
}


bool checkBoolArray(bool *daVis, int length)
{
  int i = 0;
  while (i < length)
  {
    if (daVis[i])
      return true;
    i++;
  }
  return false;
}

double trasformaDaArrayAIntNeg(int *sol, int length)
{
  double somma = 0;
  for (int i = (length - 1); i >= 0; i--)
  {
    if (sol[i] == -1)
    {
      somma += pow(2, (length - 1) - i);
    }
  }
  //cout<<"Valore negativo "<<somma<<endl;
  return somma;
}

double trasformaDaArrayAIntPos(int *sol, int length)
{
  double somma = 0;
  for (int i = (length - 1); i >= 0; i--)
  {
    if (sol[i] == 1)
    {
      somma += pow(2, (length - 1) - i);
    }
  }
  //cout<<"Valore positivo "<<somma<<endl;
  return somma;
}

void trasformaDaArrayAArray(int *sol, int length, int *temp)
{
  for (int i = 0; i < length; i++)
  {
    if (sol[i] == 1)
      sol[i] = 1;
    if (temp[i] == 1)
      sol[i] = 1;
  }
}

void trasformaDaIntAArrayPos(int *sol, int length, int val)
{
  for (int i = (length - 1); i >= 0; i--)
  {
    if (val > 0)
    {
      sol[i] = val % 2;
      val = val / 2;
    }
  }
}

void trasformaDaIntAArrayNeg(int *sol, int length, int val)
{
  for (int i = (length - 1); i >= 0; i--)
  {
    if (val > 0)
    {
      sol[i] = val % 2;
      val = val / 2;
      sol[i] = sol[i]*(-1);
    }
  }
}



__global__ void checkRow(bool *d_matrix, int *d_sol, int length, int *d_posizione, bool *d_visitato, bool *d_esisteNeiVincoli){
  int thid = blockIdx.x * blockDim.x + threadIdx.x;
  //int thid = 0;
    //printf("Sono thid %d e ho valore %d stato visitato? %d\n", thid, d_sol[thid], d_visitato[thid]);
  if(thid < length){
    if(d_sol[thid] == -1 && d_visitato[thid] == 0){
      for(int i = 0; i < length; i++){
        if(d_matrix[thid*length+i] == 1){
          if(d_sol[i] == 0){
            d_sol[i] = 1;
            if(i >= (length/2)){
              if(d_sol[i-(length/2)] == 1){
                d_posizione[1] = d_posizione[1] || 1;
              }
            }else if(i < (length/2)){
              if(d_sol[i+(length/2)] == 1){
                d_posizione[1] = d_posizione[1] || 1;
              }
            }
            d_posizione[0] = d_posizione[0] || 1;
          }
          if(d_sol[i] == -1){
            //printf("Questa soluzione non va bene\n");
            d_posizione[1] = d_posizione[1] || 1;
          }
          // printf("Trovato\n");
        }
      }
      d_visitato[thid] = 1;
      if(thid < (length/2))
        d_visitato[thid+(length/2)] = 1;
    }
  }
  __syncthreads();
}

#include <memory>
void printBool(bool *array, int length){
  
  for(int i = 0; i < length; i++){
    cout<<array[i]<<" ";
  }
  cout<<endl;
}
void printInt(int *array, int length){
  
  for(int i = 0; i < length; i++){
    cout<<array[i]<<" ";
  }
  cout<<endl;
}


#include <vector>
#include <algorithm>
bool cheK(std::vector<int> &sol){
  return std::any_of(sol.begin(), sol.end(), [](const int &i){return i == 0;});
}


int tras(int number, int let)
{
  if (number < 0)
  {
    number = abs(number) + let;
  }
  return number - 1;
}

bool checkIfSolZero(int *sol, int nTotLet)
{
  for (int i = 0; i < nTotLet; i++)
  {
    if (sol[i] == 0)
    {
      return true;
    }
  }
  return true;
}

__global__ void checkSolution(int *d_sol, int length, int * d_posizione, bool *d_matrix){
  int thid = blockIdx.x * blockDim.x + threadIdx.x;
  if(thid < length){
    //printf("Sono %d\n", thid);
    for(int i = thid*length; i < thid*(1+length); i++){
      if(d_matrix[i] == 1 && d_sol[thid] == -1 && d_sol[i%length] == -1){
        d_posizione[1] = d_posizione[1] || 1;
      }
    } 
  }

}

__global__ void completaSol(int *d_sol, int length, bool *d_esisteNeiVincoli)
{
  int thid = blockIdx.x * blockDim.x + threadIdx.x;
  //int thid = 0;
  if (thid < (length / 2))
  {
    if (d_sol[thid] == 0 && d_sol[thid + (length / 2)] != 0 && d_esisteNeiVincoli[thid] == 1)
    {
      if (d_sol[thid + (length / 2)] == 1)
        d_sol[thid] = -1;
      else if (d_sol[thid + (length / 2)] == -1)
        d_sol[thid] = 1;
    }
    if (d_sol[thid] != 0 && d_sol[thid + (length / 2)] == 0 && d_esisteNeiVincoli[thid + (length / 2)] == 1)
    {
      if (d_sol[thid] == 1)
        d_sol[thid + (length / 2)] = -1;
      else if (d_sol[thid] == -1)
        d_sol[thid + (length / 2)] = 1;
    }
  }
__syncthreads();
}

__global__ void salvaSoluzioneProx(int *d_solReg, int *d_sol_backup, int length, int cSol){
  int thid = blockIdx.x * blockDim.x + threadIdx.x;
  if(thid < length){
    //printf("HO COPIATO %d\n" , (cSol * length) + thid);
    d_solReg[(cSol * length) + thid] = d_sol_backup[thid];
  }
  __syncthreads();
}

__global__ void copiaSoluzioneProx(int *d_solReg, int *d_sol, int length, int cSol){
  int thid = blockIdx.x * blockDim.x + threadIdx.x;
  if(thid < length){
    d_sol[thid] = d_solReg[(cSol * length) + thid];
    //printf("HO COPIATO\n");
  }
  __syncthreads();
}

__global__ void newSolution(int *d_posizione){

}

__global__ void controlloNuovaSol(int *d_sol, int *d_solFinali, int length, int indexSol, int *d_posizione){
  int thid = blockIdx.x * blockDim.x + threadIdx.x;
  if(thid < indexSol){
    int cont = 0;
    int i = 0;
    do{
      i++;
      cont++;
    }while(i < length && d_solFinali[thid*length + i] == d_sol[i]);

    if(cont==length)
      d_posizione[2] = d_posizione[2] || 1;
  }
  __syncthreads();
}