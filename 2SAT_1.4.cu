#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <fstream>
#include <bits/stdc++.h>
#include <numeric> 
#include <hip/hip_runtime.h>
#include <omp.h>
#include <malloc.h>
#include <unistd.h>

using namespace std;

void funcRead(string str[]){
  string myText;
  int p = 0;
  ifstream MyReadFile("vincoli.txt");
  while (getline (MyReadFile, myText)) {
    str[p] = myText;
    p++;
  }
  MyReadFile.close();
}

string firstLine(){
  ifstream infile("vincoli.txt");
  string sLine;
  if (infile.good()){
    getline(infile, sLine);
  }
  return sLine;
}

int tras(int number, int let){
  if(number < 0){
    number = abs(number)+let;
  }
  return number-1;
}

__device__ void sistema(bool *d_matrix, bool *d_matrix2, int length, int thid){
  if(d_matrix2[thid]){
    d_matrix[thid]=1;
  }
}

__device__ void sistema2(bool *d_matrix, bool *d_matrix2, int length, int thid){
  d_matrix[thid]=d_matrix2[thid];
}

__device__ void diagonale(bool *d_matrix, int length, int thid){
  int secondo = (thid%length); 
  int primo = floorf(thid/length);
  d_matrix[(primo*length)+secondo] |= d_matrix[(secondo*length)+primo];
}

__global__ void prova(bool *d_matrix, int length, long int lengthx2, bool *d_matrix2, bool *d_matrix3){
  int thid2 = blockIdx.x * blockDim.x + threadIdx.x;
  int thid = 0;

  for(int Pass=0; Pass<ceilf((lengthx2/(blockDim.x*gridDim.x)))+1; Pass++){
    thid = thid2 + Pass*(gridDim.x*blockDim.x);

    if(thid < (lengthx2)){
      if(d_matrix[thid]){  //thid = 4
        int secondo = (thid%length);  // 4
        int primo = floorf(thid/length);  // 0
        for(int i = (secondo*length); i < ((secondo+1)*length); i++){   //da 24 
          if(d_matrix[i] && ((i%length)+1) != (primo+1)){    //24 però 24%6+1 == 1 quindi non entro
            int posizione = (primo*length) + (i%length);
            d_matrix2[posizione] = 1;
          }
        }
        int terzo = 0;
        if(secondo >= (length/2)){
          terzo = secondo - (length/2);
        }else{
          terzo = secondo + (length/2);
        }
        for(int i = (terzo*length); i < ((terzo+1)*length); i++){   //da 24
          if(d_matrix[i] && ((i%length)+1) != (primo+1)){    //24 però 24%6+1 == 1 quindi non entro
            int posizione = (primo*length) + (i%length);
            d_matrix2[posizione] = 1;
          }
        }
      }
      sistema(d_matrix3, d_matrix2, length, thid);
      sistema2(d_matrix, d_matrix2, length, thid);
    }
    __syncthreads();

    if(thid < (length*length)){
      diagonale(d_matrix3, length, thid);
    }
  }
  __syncthreads();
}

//LA DIAGONALE NON MI TORNA 
__global__ void checkDiagonale(bool *matrix, int length){
  int thid = blockIdx.x * blockDim.x + threadIdx.x;
  int thidCheck1 = 0;
  int thidCheck2 = 0;
  if(thid%(length+1) == 0 && thid < (length*length)){
    thidCheck1 = thid;
    thidCheck2 = (length+1)*(length/2) + thid;
    if(matrix[thidCheck1] == 1 && matrix[thidCheck1] == matrix[thidCheck2]){
      printf("Nella posizione %d e nella posizione %d hanno entrambi 1\n", thidCheck1, thidCheck2);
      printf("Quindi teoricamente non ci sono soluzioni\n\n");
    }
  }
  if(thid == 0)
    printf("Ho controllato la diagonale\n");
  __syncthreads();  
}

__global__ void daVisitare(bool *matrix, int *d_daVis, int length, int index){ //length è nLet tutto per due
  int thid2 = blockIdx.x * blockDim.x + threadIdx.x;
  int thid = 0;

  for(int Pass=0; Pass<ceilf(((length/2)/(blockDim.x*gridDim.x)))+1; Pass++){
    thid = thid2 + Pass*(gridDim.x*blockDim.x);

    if(thid < (length/2)){
      if(matrix[thid+index*(length/2)])
        d_daVis[thid] = thid+index*(length/2);
    }
  }
  __syncthreads();
}

__global__ void sistemaArray(int *d_daVis, int *d_temp, int length){ //length è nLet tutto per due
  int thid2 = blockIdx.x * blockDim.x + threadIdx.x;
  int thid = 0;

  for(int Pass=0; Pass<ceilf(((length/2)/(blockDim.x*gridDim.x)))+1; Pass++){
    thid = thid2 + Pass*(gridDim.x*blockDim.x);

    if(thid < length){
      if(d_temp[thid] != 0)
        d_daVis[thid] = d_temp[thid];
    }
  }
  __syncthreads();
}

bool checkBoolArray(bool *daVis, int length){
  int i = 0;
  while(i < length){
    if(daVis[i])
      return true;
    i++;
  }

  int j = 0;
  int Posto = 0;
  while(Posto < length){
    if(daVis[Posto] != 0){
      daVis[j] = daVis[Posto];
      j++;
    }
    i++;
  }
  return false;
}

__global__ void completaSol(int *d_sol, int length){
  int thid2 = blockIdx.x * blockDim.x + threadIdx.x;
  int thid = 0;

  for(int Pass=0; Pass<ceilf(((length/2)/(blockDim.x*gridDim.x)))+1; Pass++){
    thid = thid2 + Pass*(gridDim.x*blockDim.x);
    if(thid < (length/2)){
      if(d_sol[thid] == 0 && d_sol[thid+(length/2)] != 0){
        if(d_sol[thid+(length/2)] == 1){
          d_sol[thid] = -1;
        }else if(d_sol[thid+(length/2)] == -1){
          d_sol[thid] = 1;
        }
      }
      if(d_sol[thid] != 0 && d_sol[thid+(length/2)] == 0){
        if(d_sol[thid] == 1){
          d_sol[thid+(length/2)] = -1;
        }else if(d_sol[thid] == -1){
          d_sol[thid+(length/2)] = 1;
        }
      }
    }
  }
  __syncthreads();
}

__global__ void workVisit(bool *matrix, int *d_sol, int *d_daVis, int index, int length, int *posizione, int *d_temp){
  int valore = d_sol[index];
  d_daVis[index] = 0;
  int thid2 = blockIdx.x * blockDim.x + threadIdx.x;
  int thid = 0;

  for(int Pass=0; Pass<ceilf((length/(blockDim.x*gridDim.x)))+1; Pass++){
    thid = thid2 + Pass*(gridDim.x*blockDim.x);
    //thid = thid + (index*length);
    if(thid < length && d_daVis[thid] != 0)
    {  
      //printf("Io sono %d\n", thid); // qua c'è qualcosa
      if(d_sol[d_daVis[thid]] == 0 && valore == -1){
        d_temp[thid] = d_daVis[thid];
        d_sol[d_daVis[thid]] = 1;
      }
      else if(d_sol[d_daVis[thid]] == 0 && valore == 1)
      {
        d_temp[thid] = d_daVis[thid];
        d_sol[d_daVis[thid]] = 0;  //capire cosa fare qua, dovrei diramare ? -1
      }
      else if(d_sol[d_daVis[thid]] == ((-1)*valore))
      {
        d_temp[thid] = d_daVis[thid];
      }
      else if(d_sol[d_daVis[thid]] == valore && valore == -1)
      {
        printf("Impossibile ottenere una soluzione grazie a %d\n", thid2);
        atomicAdd(posizione, 1); 
      }
    }
    __syncthreads();
  }

  __syncthreads();
}


int main(void)
{ 
  string s = firstLine();
  string arrayyy[4];
  stringstream ss(s);
  string word;
  int i=0;
  while (ss >> word) {
    arrayyy[i]=word;
    i++;
  }

  int letterali = stoi(arrayyy[2]);
  int vincoli = stoi(arrayyy[3]);
  int nTotLet = (letterali*2);
  long int nTotLetx2 = nTotLet * nTotLet;
  bool matrix[nTotLetx2] = {0};
  string str[vincoli+1];
  funcRead(str);

  // #pragma omp parallel shared(str, adj)
  // { 
  //  #pragma omp for schedule(auto)
  for(int i=1; i<=vincoli; i++){
    stringstream ss(str[i]);
    string word;
    int pos = 0;
    int pos1 = 0;
    int j = 0;
    while (ss >> word && j<=1) {
      if(j==0){
        pos = tras(stoi(word), letterali);
      }
      else if(j == 1){
        pos1 = tras(stoi(word), letterali);
        matrix[((pos*nTotLet)+pos1)] = 1;
        matrix[((pos1*nTotLet)+pos)] = 1;
      }
      j++;
    }
  }

  for(int i=0; i<nTotLetx2; i++){
    cout<<matrix[i]<<" ";
    if(i%nTotLet == (nTotLet-1))
      cout<<endl;
  }
  cout<<endl;
  //}
  
  //https://docs.nvidia.com/cuda/cusparse/index.html#coo-format
  
  bool *d_matrix;
  bool *d_matrix2;
  bool *d_matrix3;

  hipMalloc(&d_matrix, nTotLetx2*sizeof(bool));
  hipMalloc(&d_matrix2, nTotLetx2*sizeof(bool));
  hipMalloc(&d_matrix3, nTotLetx2*sizeof(bool));

  hipMemcpy(d_matrix, matrix, nTotLetx2*sizeof(bool), hipMemcpyHostToDevice);
  hipMemcpy(d_matrix3, matrix, nTotLetx2*sizeof(bool), hipMemcpyHostToDevice);
  sleep(10);
  //bool out[nTotLet];
  //bool *d_out;
  //hipMalloc(&d_out, nTotLetx2*sizeof(bool));
  prova<<<40, 1024>>>(d_matrix, nTotLet, nTotLetx2, d_matrix2, d_matrix3);
  hipFree(d_matrix);
  hipFree(d_matrix2);
  checkDiagonale<<<40, 1024>>>(d_matrix3, nTotLet);


  //PROVIAMO A CERCARE UNA SOLUZIONE
  int sol[nTotLet] = {0};
  int *d_sol;
  hipMalloc(&d_sol, nTotLet*sizeof(int));
  int daVis[nTotLet] = {0};
  int *d_daVis;
  hipMalloc(&d_daVis, nTotLet*sizeof(int));
  //hipMemcpy(d_daVis, daVis, nTotLet*sizeof(int), hipMemcpyHostToDevice);
  int posizione = 0;
  int *d_posizione;
  hipMalloc(&d_posizione, sizeof(int));

  int temp[nTotLet] = {0};
  bool alreadyC = true;
  int *d_temp;
  hipMalloc(&d_temp, nTotLet*sizeof(int));
  for(int i = 0; i< letterali; i++){
    hipMemcpy(d_temp, temp, nTotLet*sizeof(int), hipMemcpyHostToDevice);
    cout<<i<<endl;
    if(!alreadyC){
      hipMemcpy(sol, d_sol, nTotLet*sizeof(int), hipMemcpyDeviceToHost);
      alreadyC = true;
    } 
    if(sol[i]==0 && sol[i+letterali]==0){
      sol[i] = -1;
      sol[i+letterali] = 1;
      daVisitare<<<40, 1024>>>(d_matrix3, d_daVis, nTotLet, i);
      hipMemcpy(daVis, d_daVis, nTotLet*sizeof(int), hipMemcpyDeviceToHost);

      int j = 0;
      int Posto = 0;
      while(Posto < nTotLet){
        if(daVis[Posto] != 0){
          daVis[j] = daVis[Posto];
          j++;
        }
        i++;
      }
      hipMemcpy(d_daVis, daVis, nTotLet*sizeof(int), hipMemcpyHostToDevice);
      hipMemcpy(d_sol, sol, nTotLet*sizeof(int), hipMemcpyHostToDevice);

      int ind = 0;
      while(ind < nTotLet && daVis[0] != 0){
        if(daVis[ind] != 0){
          workVisit<<<40, 1024>>>(d_matrix3, d_sol, d_daVis, ind, nTotLet, d_posizione, d_temp);  //posizione da cui sono partito e valore che possiede
          sistemaArray<<<40, 1024>>>(d_daVis, d_temp, nTotLet);
          hipMemcpy(daVis, d_daVis, nTotLet*sizeof(bool), hipMemcpyDeviceToHost);

        }
        if(ind == nTotLet)
          ind = 0;
        ind++;
      }  

      completaSol<<<40, 1024>>>(d_sol, nTotLet);
      alreadyC = false;
    }
  }
  //hipMemcpy(posizione, &d_posizione, sizeof(int), hipMemcpyDeviceToHost);
  cout<<"Ci sono: "<<posizione<<" discrepanze"<<endl;

  hipMemcpy(sol, d_sol, nTotLet*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(&matrix, d_matrix3, nTotLetx2*sizeof(bool), hipMemcpyDeviceToHost);




  //hipMemcpy(&out, d_out, nTotLet*sizeof(int), hipMemcpyDeviceToHost);
  
  cout<<endl;
  for(int i=0; i<nTotLetx2; i++){
    cout<<matrix[i]<<" ";
    if(i%nTotLet == (nTotLet-1))
      cout<<endl;
  }
  cout<<endl;
  //}
  cout<<"Soluzione: "<<endl;
  for(int i=0; i<nTotLet; i++){
    cout<<sol[i]<<endl;
  }
  cout<<endl;

  
  hipFree(d_temp);
  hipFree(d_matrix3);
  hipFree(d_daVis);
  hipFree(d_sol);
  free(temp);
  free(matrix);
  free(daVis);
  return 0;
}




