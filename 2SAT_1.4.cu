#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <fstream>
#include <bits/stdc++.h>
#include <numeric> 
#include <hip/hip_runtime.h>
#include <omp.h>
#include <malloc.h>
#include <unistd.h>

using namespace std;

void funcRead(string str[]){
  string myText;
  int p = 0;
  ifstream MyReadFile("vincoli.txt");
  while (getline (MyReadFile, myText)) {
    str[p] = myText;
    p++;
  }
  MyReadFile.close();
}

string firstLine(){
  ifstream infile("vincoli.txt");
  string sLine;
  if (infile.good()){
    getline(infile, sLine);
  }
  return sLine;
}

int tras(int number, int let){
  if(number < 0){
    number = abs(number)+let;
  }
  return number-1;
}

__device__ void sistema(bool *d_matrix, bool *d_matrix2, int length, int thid){
  if(d_matrix2[thid]){
    d_matrix[thid]=1;
  }
}

__device__ void sistema2(bool *d_matrix, bool *d_matrix2, int length, int thid){
  d_matrix[thid]=d_matrix2[thid];
}

__device__ void diagonale(bool *d_matrix, int length, int thid){
  int secondo = (thid%length); 
  int primo = floorf(thid/length);
  d_matrix[(primo*length)+secondo] |= d_matrix[(secondo*length)+primo];
}

__global__ void prova(bool *d_matrix, int length, long int lengthx2, bool *d_matrix2, bool *d_matrix3){
  int thid2 = blockIdx.x * blockDim.x + threadIdx.x;
  int thid = 0;

  for(int Pass=0; Pass<ceilf((lengthx2/(blockDim.x*gridDim.x)))+1; Pass++){
    thid = thid2 + Pass*(gridDim.x*blockDim.x);

    if(thid < (lengthx2)){
      if(d_matrix[thid]){  //thid = 4
        int secondo = (thid%length);  // 4
        int primo = floorf(thid/length);  // 0
        for(int i = (secondo*length); i < ((secondo+1)*length); i++){   //da 24 
          if(d_matrix[i] && ((i%length)+1) != (primo+1)){    //24 però 24%6+1 == 1 quindi non entro
            int posizione = (primo*length) + (i%length);
            d_matrix2[posizione] = 1;
          }
        }
        int terzo = 0;
        if(secondo >= (length/2)){
          terzo = secondo - (length/2);
        }else{
          terzo = secondo + (length/2);
        }
        for(int i = (terzo*length); i < ((terzo+1)*length); i++){   //da 24
          if(d_matrix[i] && ((i%length)+1) != (primo+1)){    //24 però 24%6+1 == 1 quindi non entro
            int posizione = (primo*length) + (i%length);
            d_matrix2[posizione] = 1;
          }
        }
      }
      sistema(d_matrix3, d_matrix2, length, thid);
      sistema2(d_matrix, d_matrix2, length, thid);
    }
    __syncthreads();

    if(thid < (length*length)){
      diagonale(d_matrix3, length, thid);
    }
  }
  __syncthreads();
}
//LA DIAGONALE NON MI TORNA 
__global__ void checkDiagonale(bool *matrix, int length){
  int thid2 = blockIdx.x * blockDim.x + threadIdx.x;
  int thidCheck1 = 0;
  int thidCheck2 = 0;
  //printf("%d\n", thid2);
  if(thid2%(length+1) == 0 && thid2 < (length*length)){
    thidCheck1 = thid2;
    thidCheck2 = (length+1)*(length/2) + thid2;
    if(matrix[thidCheck1] == 1 && matrix[thidCheck1] == matrix[thidCheck2]){
      printf("Nella posizione %d e nella posizione %d hanno entrambi 1\n", thidCheck1, thidCheck2);
      printf("Quindi teoricamente non ci sono soluzioni\n\n");
    }
  }
  if(thid2 == 0)
   printf("Ho controllato la diagonale\n");
  __syncthreads();  
}

__global__ void findSolution(bool *matrix, int *soluzione, int length, int bit){ //length è nLet tutto per due
  //il bit sarà quello messo a vero (tipo -1)
  int thid = blockIdx.x * blockDim.x + threadIdx.x;
  int lengthDiv2 = length/2;
  if(thid == 0){
    if(bit < 0){
      soluzione[abs(bit)+lengthDiv2-1] = 1;
      soluzione[abs(bit)-1] = -1;
    }else{
      soluzione[bit-1] = 1;
      soluzione[bit+lengthDiv2-1] = -1;
    }
  }
  __syncthreads();
}


int main(void)
{ 
  string s = firstLine();
  string arrayyy[4];
  stringstream ss(s);
  string word;
  int i=0;
  while (ss >> word) {
    arrayyy[i]=word;
    i++;
  }

  int letterali = stoi(arrayyy[2]);
  int vincoli = stoi(arrayyy[3]);
  int nTotLet = (letterali*2);
  long int nTotLetx2 = nTotLet * nTotLet;
  bool matrix[nTotLetx2];
  string str[vincoli+1];
  funcRead(str);

  // #pragma omp parallel shared(str, adj)
  // { 
  //  #pragma omp for schedule(auto)
  for(int i=1; i<=vincoli; i++){
    stringstream ss(str[i]);
    string word;
    int pos = 0;
    int pos1 = 0;
    int j = 0;
    while (ss >> word && j<=1) {
      if(j==0){
        pos = tras(stoi(word), letterali);
      }
      else if(j == 1){
        pos1 = tras(stoi(word), letterali);
        matrix[((pos*nTotLet)+pos1)] = 1;
        matrix[((pos1*nTotLet)+pos)] = 1;
      }
      j++;
    }
  }

  /*for(int i=0; i<nTotLetx2; i++){
    cout<<matrix[i]<<" ";
    if(i%nTotLet == (nTotLet-1))
      cout<<endl;
  }
  cout<<endl;
  //}*/
  
  //https://docs.nvidia.com/cuda/cusparse/index.html#coo-format
  
  bool *d_matrix;
  bool *d_matrix2;
  bool *d_matrix3;

  hipMalloc(&d_matrix, nTotLetx2*sizeof(bool));
  hipMalloc(&d_matrix2, nTotLetx2*sizeof(bool));
  hipMalloc(&d_matrix3, nTotLetx2*sizeof(bool));

  hipMemcpy(d_matrix, matrix, nTotLetx2*sizeof(bool), hipMemcpyHostToDevice);
  hipMemcpy(d_matrix3, matrix, nTotLetx2*sizeof(bool), hipMemcpyHostToDevice);
  sleep(10);
  //bool out[nTotLet];
  //bool *d_out;
  //hipMalloc(&d_out, nTotLetx2*sizeof(bool));
  prova<<<40, 1024>>>(d_matrix, nTotLet, nTotLetx2, d_matrix2, d_matrix3);
  hipFree(d_matrix);
  hipFree(d_matrix2);
  checkDiagonale<<<40, 1024>>>(d_matrix3, nTotLet);


  //PROVIAMO A CERCARE UNA SOLUZIONE
  int soluzione[nTotLet] = {0};
  int *d_soluzione;
  int bit = 1;
  hipMalloc(&d_soluzione, nTotLet*sizeof(int));
  hipMemcpy(d_soluzione, soluzione, nTotLet*sizeof(int), hipMemcpyHostToDevice);
  //findSolution<<<40, 1024>>>(d_matrix3, d_soluzione, nTotLet, bit);




  hipMemcpy(&matrix, d_matrix3, nTotLetx2*sizeof(bool), hipMemcpyDeviceToHost);




  //hipMemcpy(&out, d_out, nTotLet*sizeof(int), hipMemcpyDeviceToHost);
  

  /*for(int i=0; i<nTotLetx2; i++){
    cout<<matrix[i]<<" ";
    if(i%nTotLet == (nTotLet-1))
      cout<<endl;
  }
  cout<<endl;
  //}*/

  hipFree(d_matrix3);
  return 0;
}




