#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <fstream>
#include <bits/stdc++.h>
#include <numeric>
#include <hip/hip_runtime.h>
#include <omp.h>
#include <malloc.h>
#include <unistd.h>
#include <list>
#include <bits/stdc++.h>

using namespace std;

void funcRead(string str[])
{
  string myText;
  int p = 0;
  ifstream MyReadFile("vincoli.txt");
  while (getline(MyReadFile, myText))
  {
    str[p] = myText;
    p++;
  }
  MyReadFile.close();
}

string firstLine()
{
  ifstream infile("vincoli.txt");
  string sLine;
  if (infile.good())
  {
    getline(infile, sLine);
  }
  return sLine;
}

__device__ void diagonale(bool *d_matrix, int length, int thid)
{
  int secondo = (thid % length);
  int primo = floorf(thid / length);
  d_matrix[(primo * length) + secondo] |= d_matrix[(secondo * length) + primo];
}

__global__ void prova(bool *d_matrix, int length, long int lengthx2, int *d_posizione)
{
  int thid2 = blockIdx.x * blockDim.x + threadIdx.x;
  int thid = 0;
  /*if(thid2 == 0){
    printf("Ma quante volte rientro\n");
  }*/

  for (int Pass = 0; Pass < ceilf((lengthx2 / (blockDim.x * gridDim.x))) + 1; Pass++)
  {
    thid = thid2 + Pass * (gridDim.x * blockDim.x);

    if (thid < (lengthx2))
    {
      if (d_matrix[thid])
      { // thid = 4
        int secondo = (thid % length);
        if (secondo >= (length / 2))
          secondo = secondo - (length / 2);
        else
          secondo = secondo + (length / 2);                                 // 4
        int primo = floorf(thid / length); // 0
        for (int i = (secondo * length); i < ((secondo + 1) * length); i++)
        { // da 24
          if (d_matrix[i] && ((i % length) + 1) != (primo + 1))
          { // 24 però 24%6+1 == 1 quindi non entro
            int posizione = (primo * length) + (i % length);
            d_matrix[posizione] = 1;
            d_posizione[0] = 1;
          }
        }
      }
    }
    __syncthreads();

    if (thid < (length * length))
    {
      diagonale(d_matrix, length, thid);
    }
  }
  __syncthreads();
}

// LA DIAGONALE NON MI TORNA
__global__ void checkDiagonale(bool *matrix, int length)
{
  int thid = blockIdx.x * blockDim.x + threadIdx.x;
  int thidCheck1 = 0;
  int thidCheck2 = 0;
  if (thid % (length + 1) == 0 && thid < (length * length))
  {
    thidCheck1 = thid;
    thidCheck2 = (length + 1) * (length / 2) + thid;
    if (matrix[thidCheck1] == 1 && matrix[thidCheck1] == matrix[thidCheck2])
    {
      printf("Nella posizione %d e nella posizione %d hanno entrambi 1\n", thidCheck1, thidCheck2);
      printf("Quindi teoricamente non ci sono soluzioni\n\n");
    }
  }
  if (thid == 0)
    printf("Ho controllato la diagonale\n");
  __syncthreads();
}


bool checkBoolArray(bool *daVis, int length)
{
  int i = 0;
  while (i < length)
  {
    if (daVis[i])
      return true;
    i++;
  }
  return false;
}

double trasformaDaArrayAIntNeg(int *sol, int length)
{
  double somma = 0;
  for (int i = (length - 1); i >= 0; i--)
  {
    if (sol[i] == -1)
    {
      somma += pow(2, (length - 1) - i);
    }
  }
  //cout<<"Valore negativo "<<somma<<endl;
  return somma;
}

double trasformaDaArrayAIntPos(int *sol, int length)
{
  double somma = 0;
  for (int i = (length - 1); i >= 0; i--)
  {
    if (sol[i] == 1)
    {
      somma += pow(2, (length - 1) - i);
    }
  }
  //cout<<"Valore positivo "<<somma<<endl;
  return somma;
}

void trasformaDaArrayAArray(int *sol, int length, int *temp)
{
  for (int i = 0; i < length; i++)
  {
    if (sol[i] == 1)
      sol[i] = 1;
    if (temp[i] == 1)
      sol[i] = 1;
  }
}

void trasformaDaIntAArrayPos(int *sol, int length, int val)
{
  for (int i = (length - 1); i >= 0; i--)
  {
    if (val > 0)
    {
      sol[i] = val % 2;
      val = val / 2;
    }
  }
}

void trasformaDaIntAArrayNeg(int *sol, int length, int val)
{
  for (int i = (length - 1); i >= 0; i--)
  {
    if (val > 0)
    {
      sol[i] = val % 2;
      val = val / 2;
      sol[i] = sol[i]*(-1);
    }
  }
}



__global__ void checkRow(bool *d_matrix, int *d_sol, int length, int *d_posizione, bool *d_visitato, bool *d_esisteNeiVincoli){
  int thid = blockIdx.x * blockDim.x + threadIdx.x;
  //int thid = 0;
    //printf("Sono thid %d e ho valore %d stato visitato? %d\n", thid, d_sol[thid], d_visitato[thid]);
  if(thid < length){
    if(d_sol[thid] == -1 && d_visitato[thid] == 0){
      for(int i = 0; i < length; i++){
        if(d_matrix[thid*length+i] == 1){
          if(d_sol[i] == 0){
            d_sol[i] = 1;
            if(i >= (length/2)){
              if(d_sol[i-(length/2)] == 1){
                d_posizione[1] = d_posizione[1] || 1;
              }
            }else if(i < (length/2)){
              if(d_sol[i+(length/2)] == 1){
                d_posizione[1] = d_posizione[1] || 1;
              }
            }
            d_posizione[0] = d_posizione[0] || 1;
          }
          if(d_sol[i] == -1){
            //printf("Questa soluzione non va bene\n");
            d_posizione[1] = d_posizione[1] || 1;
          }
          // printf("Trovato\n");
        }
      }
      d_visitato[thid] = 1;
      if(thid < (length/2))
        d_visitato[thid+(length/2)] = 1;
    }
  }
  __syncthreads();
}

#include <memory>
void printBool(bool *array, int length){
  
  for(int i = 0; i < length; i++){
    cout<<array[i]<<" ";
  }
  cout<<endl;
}
void printInt(int *array, int length){
  
  for(int i = 0; i < length; i++){
    cout<<array[i]<<" ";
  }
  cout<<endl;
}


#include <vector>
#include <algorithm>
bool cheK(std::vector<int> &sol){
  return std::any_of(sol.begin(), sol.end(), [](const int &i){return i == 0;});
}


int tras(int number, int let)
{
  if (number < 0)
  {
    number = abs(number) + let;
  }
  return number - 1;
}

bool checkIfSolZero(int *sol, int nTotLet)
{
  for (int i = 0; i < nTotLet; i++)
  {
    if (sol[i] == 0)
    {
      return true;
    }
  }
  return true;
}

__global__ void checkSolution(int *d_sol, int length, int * d_posizione, bool *d_matrix){
  int thid = blockIdx.x * blockDim.x + threadIdx.x;
  if(thid < length){
    //printf("Sono %d\n", thid);
    for(int i = thid*length; i < thid*(1+length); i++){
      if(d_matrix[i] == 1 && d_sol[thid] == -1 && d_sol[i%length] == -1){
        d_posizione[1] = d_posizione[1] || 1;
      }
    } 
  }

}

__global__ void completaSol(int *d_sol, int length, bool *d_esisteNeiVincoli)
{
  int thid = blockIdx.x * blockDim.x + threadIdx.x;
  //int thid = 0;
  if (thid < (length / 2))
  {
    if (d_sol[thid] == 0 && d_sol[thid + (length / 2)] != 0 && d_esisteNeiVincoli[thid] == 1)
    {
      if (d_sol[thid + (length / 2)] == 1)
        d_sol[thid] = -1;
      else if (d_sol[thid + (length / 2)] == -1)
        d_sol[thid] = 1;
    }
    if (d_sol[thid] != 0 && d_sol[thid + (length / 2)] == 0 && d_esisteNeiVincoli[thid + (length / 2)] == 1)
    {
      if (d_sol[thid] == 1)
        d_sol[thid + (length / 2)] = -1;
      else if (d_sol[thid] == -1)
        d_sol[thid + (length / 2)] = 1;
    }
  }
__syncthreads();
}

__global__ void salvaSoluzioneProx(int *d_solReg, int *d_sol_backup, int length, int cSol){
  int thid = blockIdx.x * blockDim.x + threadIdx.x;
  if(thid < length){
    //printf("HO COPIATO %d\n" , (cSol * length) + thid);
    d_solReg[(cSol * length) + thid] = d_sol_backup[thid];
  }
  __syncthreads();
}

__global__ void copiaSoluzioneProx(int *d_solReg, int *d_sol, int length, int cSol){
  int thid = blockIdx.x * blockDim.x + threadIdx.x;
  if(thid < length){
    d_sol[thid] = d_solReg[(cSol * length) + thid];
    //printf("HO COPIATO\n");
  }
  __syncthreads();
}

__global__ void newSolution(int *d_posizione){

}

__global__ void controlloNuovaSol(int *d_sol, int *d_solFinali, int length, int indexSol, int *d_posizione){
  int thid = blockIdx.x * blockDim.x + threadIdx.x;
  if(thid < indexSol){
    int cont = 0;
    int i = 0;
    do{
      i++;
      cont++;
    }while(i < length && d_solFinali[thid*length + i] == d_sol[i]);

    if(cont==length)
      d_posizione[2] = d_posizione[2] || 1;
  }
  __syncthreads();
}



int main(void)
{
  string s = firstLine();
  string arrayyy[4];
  stringstream ss(s);
  string word;
  int i = 0;
  while (ss >> word)
  {
    arrayyy[i] = word;
    i++;
  }

  int letterali = stoi(arrayyy[2]);
  int vincoli = stoi(arrayyy[3]);
  int nTotLet = (letterali * 2);
  long int nTotLetx2 = nTotLet * nTotLet;
  bool matrix[nTotLetx2] = {0};
  string str[vincoli + 1];
  funcRead(str);
  
  bool esisteNeiVincoli[nTotLet] = {false};
  // #pragma omp parallel shared(str, adj)
  // {
  //  #pragma omp for schedule(auto)
  for (int i = 1; i <= vincoli; i++)
  {
    stringstream ss(str[i]);
    string word;
    int pos = 0;
    int pos1 = 0;
    int j = 0;
    while (ss >> word && j <= 1)
    {
      if (j == 0)
      {
        pos = tras(stoi(word), letterali);
      }
      else if (j == 1)
      {
        pos1 = tras(stoi(word), letterali);
        matrix[((pos * nTotLet) + pos1)] = 1;
        matrix[((pos1 * nTotLet) + pos)] = 1;
      }
      esisteNeiVincoli[pos] = true;
      esisteNeiVincoli[pos1] = true;
      j++;
    }
  }
  //}
  bool *d_esisteNeiVincoli;
  hipMalloc(&d_esisteNeiVincoli, nTotLet * sizeof(bool));
  hipMemcpy(d_esisteNeiVincoli, esisteNeiVincoli, nTotLet * sizeof(bool), hipMemcpyHostToDevice);

  bool *d_matrix;
  hipMalloc(&d_matrix, nTotLetx2 * sizeof(bool));
  hipMemcpy(d_matrix, matrix, nTotLetx2 * sizeof(bool), hipMemcpyHostToDevice);

  hipDeviceSynchronize();

  int posizione[3] = {0};
  int *d_posizione;
  hipMalloc(&d_posizione, 3 * sizeof(int));

  prova<<<40, 1024>>>(d_matrix, nTotLet, nTotLetx2, d_posizione);
  hipDeviceSynchronize();
  prova<<<40, 1024>>>(d_matrix, nTotLet, nTotLetx2, d_posizione);
  hipDeviceSynchronize();

      
  posizione[0] = 0;
  hipMemcpy(d_posizione, posizione, 3 * sizeof(int), hipMemcpyHostToDevice);
          
  checkDiagonale<<<40, 1024>>>(d_matrix, nTotLet);

  hipDeviceSynchronize();

  int sol[nTotLet] = {0};
  int *d_sol;
  hipMalloc(&d_sol, nTotLet * sizeof(int));
  int sol_backup[nTotLet] = {0};
  int *d_sol_backup;
  hipMalloc(&d_sol_backup, nTotLet * sizeof(int));

  int k = 3;
  int indexSol = 0;
  int cSol = 0;
  list<double> prox[1];
  int solReg[nTotLet * 1000];
  int *d_solReg;
  hipMalloc(&d_solReg, (nTotLet * 1000) * sizeof(int));
  hipMemcpy(d_solReg, solReg, (nTotLet * 1000) * sizeof(int), hipMemcpyHostToDevice);

  //array per soluzioni finali
  int solFinali[nTotLet * k];
  int *d_solFinali;
  hipMalloc(&d_solFinali, (nTotLet * k) * sizeof(int));
  hipMemcpy(d_solFinali, solFinali, (nTotLet * k) * sizeof(int), hipMemcpyHostToDevice);

  bool visitato[nTotLet] = {0};
  bool *d_visitato;

  hipMalloc(&d_visitato, nTotLet * sizeof(bool));
  hipMemcpy(d_visitato, visitato, nTotLet * sizeof(bool), hipMemcpyHostToDevice);
  hipMemcpy(d_sol, sol, nTotLet * sizeof(int), hipMemcpyHostToDevice);
  bool riprendoSoluzione = false;

  i = 0;

  bool esiste = false;
  bool continua = false;
  do{
    continua = false;
    hipMemcpy(d_sol, sol, nTotLet * sizeof(int), hipMemcpyHostToDevice);
    do{
      posizione[0] = 0;
      hipMemcpy(d_posizione, posizione, 3 * sizeof(int), hipMemcpyHostToDevice);
      if(sol[i] == 0 && sol[i+letterali] == 0 && riprendoSoluzione == false){
        memcpy(sol_backup, sol, nTotLet*sizeof(int));
        if(esisteNeiVincoli[i]){
          sol[i] = -1;
          sol_backup[i] = 1;
          esiste = true;
        }
        if(esisteNeiVincoli[i + letterali]){
          sol[i + letterali] = 1;
          sol_backup[i + letterali] = -1;
          esiste = true;
        }
        if(esiste){
          prox[0].push_back(i);
          hipMemcpy(d_sol_backup, sol_backup, nTotLet * sizeof(int), hipMemcpyHostToDevice);
          //printInt(sol_backup, nTotLet);
          salvaSoluzioneProx<<<40, 1024>>>(d_solReg, d_sol_backup, nTotLet, cSol);
          //cout<<cSol<<endl; 
          hipDeviceSynchronize();
          cSol++;
          hipMemcpy(d_sol, sol, nTotLet * sizeof(int), hipMemcpyHostToDevice);
        }
      }
      if(!riprendoSoluzione){
        if(esiste){
          riprendoSoluzione = true;   
          esiste = false;       
        }
        i++;
        cout<<i<<endl;
      }
      if(riprendoSoluzione){
        checkRow<<<40, 1024>>>(d_matrix, d_sol, nTotLet, d_posizione, d_visitato, d_esisteNeiVincoli);
        hipDeviceSynchronize();
        hipMemcpy(posizione, d_posizione, 3 * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(sol, d_sol, nTotLet * sizeof(int), hipMemcpyDeviceToHost);  //Da migliorare
        if(posizione[0] == 0)
          riprendoSoluzione = false;
      }
    }while(i < letterali && posizione[1] == 0); 

    hipDeviceSynchronize();
    hipMemcpy(posizione, d_posizione, 3 * sizeof(int), hipMemcpyDeviceToHost);
    if(posizione[1] == 0){
      //cout<<"Trovata una soluzione"<<endl;
      //printInt(sol, nTotLet);
      completaSol<<<40, 1024>>>(d_sol, nTotLet, d_esisteNeiVincoli);
      hipDeviceSynchronize();
      hipMemcpy(sol, d_sol, nTotLet * sizeof(int), hipMemcpyDeviceToHost); 
      /*cout<<"Soluzione sistemata"<<endl;
      for (int ssif = 0; ssif < nTotLet; ssif++)
      {   
          if(!esisteNeiVincoli[ssif])
            cout<<-2<<" ";
          else
            cout << sol[ssif] << " ";
      }
      cout<<endl;
      cout<<endl;*/
      //SALVARE SE NUOVA
      //chiamo funzione 
      if(indexSol > 0)
        controlloNuovaSol<<<40, 1024>>>(d_sol, d_solFinali, nTotLet, indexSol, d_posizione);
      //cout<<"Valore indexaSol"<<endl;

      hipMemcpy(posizione, d_posizione, 3 * sizeof(int), hipMemcpyDeviceToHost);
      if(posizione[2] == 0 || indexSol == 0){
        k--;
        for (int ssif = 0; ssif < nTotLet; ssif++)
        {   
            solFinali[indexSol * nTotLet + ssif] = sol[ssif];
        }
        //printInt(sol, nTotLet);
        indexSol++;
        hipMemcpy(d_solFinali, solFinali, (nTotLet * k) * sizeof(int), hipMemcpyHostToDevice);
      }
     //cout<<"Array delle soluzioni"<<endl;
      //printInt(solFinali, nTotLet*(indexSol));
      posizione[2] = 0;
      hipMemcpy(d_posizione, posizione, 3 * sizeof(int), hipMemcpyHostToDevice);
    }

    posizione[1] = 0;
    hipMemcpy(d_posizione, posizione, 3 * sizeof(int), hipMemcpyHostToDevice);
    if (cSol > 0){
      memset(sol, 0, nTotLet * sizeof(int));
      i = prox[0].back();
      prox[0].pop_back();
      cSol--;
      copiaSoluzioneProx<<<40, 1024>>>(d_solReg, d_sol, nTotLet, cSol); 
      hipDeviceSynchronize();
      hipMemcpy(sol, d_sol, nTotLet * sizeof(int), hipMemcpyDeviceToHost);
      //printInt(sol, nTotLet);
      hipMemcpy(d_visitato, visitato, nTotLet * sizeof(bool), hipMemcpyHostToDevice);
      riprendoSoluzione = true;
      continua = true;
    }

  }while (continua && k > 0);

  ofstream myfile;
  myfile.open ("solution.txt");
  for(int ind = 0; ind < nTotLet*indexSol; ind++){
    myfile << solFinali[ind]<<" ";
    if(ind%nTotLet == (nTotLet-1) && ind != 0 && ind != (nTotLet*indexSol-1))
      myfile << "\n";
  }

  myfile.close();
  cout<<endl;
  cout<<"TERMINATO"<<endl;
  cout<<"k vale ora: "<<k<<endl;
  if(k == 0)
    cout<<"Ci sono tutte le soluzioni che cercavi"<<endl;
  hipFree(d_matrix);
  hipFree(d_esisteNeiVincoli);
  hipFree(d_posizione);
  hipFree(d_sol);
  hipFree(d_sol_backup);
  hipFree(d_solFinali);
  hipFree(d_solReg);
  hipFree(d_visitato);
  return 0;
}
