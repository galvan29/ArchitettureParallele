#include <stdio.h>
#include <iostream>
#include <cmath>
#include <fstream>
#include <bits/stdc++.h>
#include <numeric> 
#include <hip/hip_runtime.h>
#include <omp.h>
using namespace std;

__global__ void accumulateKernel(bool * CHECK, int len, bool * out){
  int thid = blockIdx.x * blockDim.x + threadIdx.x;
  int thidI = 0;
    
  if(thid == 0)
    *out=true;

  __syncthreads();
  
  for(int Pass=0; Pass<ceilf((len/(blockDim.x)))+1; Pass++){
    thidI = thid + Pass*(gridDim.x*blockDim.x );
    if(thidI<len){
      
      if(!CHECK[thidI] && *out){
        *out=false;
      }
    }
  }
  __syncthreads();
  
}

__global__ void trasformaDecBin(int arr[], int val, int mat[], int vincoli, bool *CHECK, int letterali){

    int thid = (blockIdx.x*blockDim.x)+threadIdx.x;
    int thidI;
    
    if(thid==0){
      for(int i=0; i<letterali; i++){ 
        if(val>0){
          arr[i]=val%2;    
          val = val/2;  
        }else{
          arr[i]=0;
        }
      } 
    }

    __syncthreads();
    for(int Pass=0; Pass<ceilf((vincoli/(gridDim.x*blockDim.x)))+1; Pass++){
      thidI = thid + Pass*(gridDim.x*blockDim.x);
      
      if(thidI<vincoli){
        int a1 = mat[thidI*3+0];
        int a2 = mat[thidI*3+1];
        if(a1 < 0){
          a1 = a1*(-1);
          if(arr[letterali-a1] == 0)
            a1 = 1;
          else
            a1 = 0;
        }
        else
          a1 = arr[letterali-a1];

        if(a2 < 0){
          a2 = a2*(-1);
          if(arr[letterali-a2] == 0)
            a2 = 1;
          else
            a2 = 0;
        }
        else
          a2 = arr[letterali-a2];

        int somma = a1 + a2;
        if(somma == 0){
          CHECK[thidI] = false;
        }else {
          CHECK[thidI] = true;
        }
      } 
    }
    __syncthreads();
}

void funcRead(string str[]){
  string myText;
  int p = 0;
  ifstream MyReadFile("vincoli.txt");
  while (getline (MyReadFile, myText)) {
    str[p] = myText;
    p++;
  }
  MyReadFile.close();
}

string firstLine(){
  ifstream infile("vincoli.txt");
  string sLine;
  if (infile.good()){
    getline(infile, sLine);
  }
  return sLine;
}

int main(void)
{
  string s = firstLine();
  string arrayyy[4];
  stringstream ss(s);
  string word;
  int i=0;
  while (ss >> word) {
    arrayyy[i]=word;
    i++;
  }
  int letterali = stoi(arrayyy[2]);
  int vincoli = stoi(arrayyy[3]);
  double valore = 0;
  //int arr[letterali] = {0}; 
  string str[vincoli+1];
  funcRead(str);
  

  int *letter, *d_letter;
  letter = (int*)malloc(letterali*sizeof(int));

  int matrice[vincoli*3];
  #pragma omp parallel shared(str, matrice)
  { 
    //cout<<omp_get_num_threads()<<endl;
    #pragma omp for schedule(auto)
    for(int i=1; i<(vincoli+1); i++){
      //cout<<"Ciao sono il thread: "<<omp_get_thread_num()<<endl;
      stringstream ss(str[i]);
      string word;
      int j = 0;

      while (ss >> word) {
        matrice[(i-1)*3+j] = stoi(word);
        j++;
      }
    }
  }

  int *d_matrice;
  hipMalloc(&d_matrice, (vincoli*3)*sizeof(int));
  hipMemcpy(d_matrice, matrice, (vincoli*3)*sizeof(int), hipMemcpyHostToDevice);
  bool *CHECK, *d_CHECK;
  CHECK = (bool*)malloc(vincoli*sizeof(bool));

  hipMalloc(&d_letter, letterali*sizeof(int)); 
  hipMalloc(&d_CHECK, (vincoli)*sizeof(bool));

  hipMemcpy(d_letter, letter, letterali*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_CHECK, CHECK, (vincoli)*sizeof(bool), hipMemcpyHostToDevice);

  bool *d_out;
  bool out;
  hipMalloc(&d_out, sizeof(bool));

  int k = 0;
  i = 0;
  double elevamento = pow(2, letterali);
  //cout<<elevamento<<endl; 
  while(k<5 && valore<elevamento)
  { 
    cout<<"Sto facendo il valore "<<valore<<endl;
    cout<<valore/elevamento<<"%"<<endl;
    trasformaDecBin<<<40, 1024>>>(d_letter, valore, d_matrice, vincoli, d_CHECK, letterali);
    hipDeviceSynchronize();
    accumulateKernel<<<40, 1024>>>(d_CHECK, vincoli, d_out);
    hipDeviceSynchronize();
    hipMemcpy(&out, d_out, sizeof(bool), hipMemcpyDeviceToHost);
    if(out){  
      cout<<endl;
      cout<<"Ok per il valore "<<valore<<endl;
      k++;
    }
    valore++;
  }
  cout<<endl<<"Numero sol trovate: "<<k<<endl;

  hipFree(d_letter);
  hipFree(d_matrice);
  hipFree(d_out);
  free(letter);

  return 0;
}




