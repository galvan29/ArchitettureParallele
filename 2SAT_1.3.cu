#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <fstream>
#include <bits/stdc++.h>
#include <numeric> 
#include <hip/hip_runtime.h>
#include <omp.h>
#include <malloc.h>
#include "BigInt.hpp"

using namespace std;

void funcRead(string str[]){
  string myText;
  int p = 0;
  ifstream MyReadFile("vincoli.txt");
  while (getline (MyReadFile, myText)) {
    str[p] = myText;
    p++;
  }
  MyReadFile.close();
}

string firstLine(){
  ifstream infile("vincoli.txt");
  string sLine;
  if (infile.good()){
    getline(infile, sLine);
  }
  return sLine;
}

int tras(int number, int let){
  if(number < 0){
    number = abs(number)+let;
  }
  return number-1;
}

__device__ void sistema(bool *d_matrix, bool *d_matrix2, int thid){
  if(d_matrix2[thid]){
    d_matrix[thid]=1;
  }
}

__device__ void sistema2(bool *d_matrix, bool *d_matrix2, BigInt length, int thid){
  d_matrix[thid]=d_matrix2[thid];
}

__device__ void diagonale(bool *d_matrix, BigInt length, int thid){
  long int secondo = (thid%length).to_long(); 
  long int primo = floorf(thid/length.to_long());
  d_matrix[(primo*length.to_long())+secondo] |= d_matrix[(secondo*length.to_long())+primo];
}

__global__ void prova(bool *d_matrix, BigInt length, BigInt lengthx2, bool *d_matrix2, bool *d_matrix3){
  int thid2 = blockIdx.x * blockDim.x + threadIdx.x;
//length numero letterali
  int thid = 0;
  
  for(int Pass=0; Pass<ceilf((lengthx2.to_long()/(blockDim.x*gridDim.x)))+1; Pass++){
    thid = thid2 + Pass*(gridDim.x*blockDim.x);

    if(thid < (lengthx2)){
      if(d_matrix[thid]){  //thid = 4
        BigInt secondo = (thid%length);  // 4
        BigInt primo = floorf(thid/length.to_long());  // 0
        for(BigInt i = (secondo*length); i < ((secondo+1)*length); i++){   //da 24
          //printf("%d %d %d\n", d_matrix[i], (primo+1), i);  
          if(d_matrix[i.to_long()] && ((i%length.to_long())+1) != (primo+1)){    //24 però 24%6+1 == 1 quindi non entro
            BigInt posizione = (primo*length) + (i%length);
            d_matrix2[posizione.to_long()] = 1;
          }
        }
        BigInt terzo = 0;
        if(secondo > (length/2)){
          terzo = secondo - (length/2);
        }else{
          terzo = secondo + (length/2);
        }
        for(BigInt i = (terzo*length); i < ((terzo+1)*length); i++){   //da 24
          //printf("%d %d %d\n", d_matrix[i], (primo+1), i);  
          if(d_matrix[i.to_long()] && ((i%length)+1) != (primo+1)){    //24 però 24%6+1 == 1 quindi non entro
            BigInt posizione = (primo*length) + (i%length);
            d_matrix2[posizione.to_long()] = 1;
          }
        }
      }
      sistema(d_matrix3, d_matrix2, thid);
      sistema2(d_matrix, d_matrix2, length, thid);
    }
    __syncthreads();

    if(thid < lengthx2){
      diagonale(d_matrix3, length, thid);
    }
  }
  __syncthreads();
}
//LA DIAGONALE NON MI TORNA
__global__ void checkDiagonale(bool *matrix, BigInt length, BigInt lengthx2){
  int thid2 = blockIdx.x * blockDim.x + threadIdx.x;
  int thidCheck1 = 0;
  int thidCheck2 = 0;
  BigInt big1 = (length+1);
  BigInt big2 = (length/2);
  big1 = big1 * big2;
  if(thid2%(length+1) == 0 && thid2 < lengthx2){
    thidCheck1 = thid2;
    thidCheck2 = big1.to_long() + thid2;
    if(matrix[thidCheck1] == 1 && matrix[thidCheck1] == matrix[thidCheck2]){
      printf("Nella posizione %d e nella posizione %d hanno entrambi 1\n", thidCheck1, thidCheck2);
      printf("Quindi teoricamente non ci sono soluzioni\n\n");
    }
  }
  if(thid2 == 0)
   printf("Ho controllato la diagonale\n");
  __syncthreads();  
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int main(void)
{ 
  string s = firstLine();
  string arrayyy[4];
  stringstream ss(s);
  string word;
  int i=0;
  while (ss >> word) {
    arrayyy[i]=word;
    i++;
  }

  BigInt letterali = stoi(arrayyy[2]);
  BigInt vincoli = stoi(arrayyy[3]);
  BigInt nTotLet = (letterali*2);
  BigInt big1 = nTotLet;
  big1 = big1 * big1;
  bool matrix[big1.to_long()];
  string str[vincoli.to_long()+1];
  funcRead(str);

  // #pragma omp parallel shared(str, adj)
  // { 
  //  #pragma omp for schedule(auto)
  for(int i=1; i<=vincoli; i++){
    stringstream ss(str[i]);
    string word;
    BigInt pos = 0;
    BigInt pos1 = 0;
    BigInt j = 0;
    while (ss >> word && j<=1) {
      if(j==0){
        pos = tras(stoi(word), letterali.to_int());
      }
      else if(j == 1){
        pos1 = tras(stoi(word), letterali.to_int());
        matrix[((pos*nTotLet)+pos1).to_long()] = 1;
        matrix[((pos1*nTotLet)+pos).to_long()] = 1;
      }
      j++;
    }
  }

  /*for(int i=0; i<nTotLetx2; i++){
    cout<<matrix[i]<<" ";
    if(i%nTotLet == (nTotLet-1))
      cout<<endl;
  }
  cout<<endl;
  //}*/
  
  //https://docs.nvidia.com/cuda/cusparse/index.html#coo-format
  
  bool *d_matrix;
  bool *d_matrix2;
  bool *d_matrix3;

  gpuErrchk(hipMalloc(&d_matrix, big1.to_long()*sizeof(bool)));
  gpuErrchk(hipMalloc(&d_matrix2, big1.to_long()*sizeof(bool)));
  gpuErrchk(hipMalloc(&d_matrix3, big1.to_long()*sizeof(bool)));
  hipDeviceSynchronize();

  hipMemcpy(d_matrix, matrix, big1.to_long()*sizeof(bool), hipMemcpyHostToDevice);
  hipMemcpy(d_matrix3, matrix, big1.to_long()*sizeof(bool), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  //bool out[nTotLet];
  //bool *d_out;
  //hipMalloc(&d_out, nTotLetx2*sizeof(bool));
  prova<<<40, 1024>>>(d_matrix, nTotLet, big1, d_matrix2, d_matrix3);
  hipDeviceSynchronize();
  hipFree(d_matrix);
  hipFree(d_matrix2);
  checkDiagonale<<<40, 1024>>>(d_matrix3, nTotLet, big1);
  hipDeviceSynchronize();
  hipMemcpy(&matrix, d_matrix3, big1.to_long()*sizeof(bool), hipMemcpyDeviceToHost);
  //hipMemcpy(&out, d_out, nTotLet*sizeof(int), hipMemcpyDeviceToHost);
  
  /*for(int i=0; i<nTotLetx2; i++){
    cout<<matrix[i]<<" ";
    if(i%nTotLet == (nTotLet-1))
      cout<<endl;
  }
  cout<<endl;
  //}*/

  return 0;
}




