#include "hip/hip_runtime.h"
#include "device.cu"
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <fstream>
#include <numeric>
#include <hip/hip_runtime.h>
#include <omp.h>
#include <malloc.h>
#include <list>
#include <string>
#include <cstdio>
#include <cstdlib>
#include <istream>
#include <sstream>
#include <algorithm>
#include <chrono>
#include <map>
#include <utility>

using namespace std;

int main(int argn, char *args[])  //main
{
	int nBlock = 40, nThread = 1024;
	bool arg1 = false;
	bool arg2 = false;
	int k = 1;  							                                                //number of solutiont o find
	string nomeFile = "vincoli/v1.txt";
	
	if(argn>1){
		for(int i=0; i<argn; i++)
		if(std::string(args[i]).substr(0,3) == "-K="){
			k = std::stoi(std::string(args[i]).substr(3));
			arg1=true;
		}
		else if(std::string(args[i]).substr(0,6) == "-file="){
			nomeFile = "vincoli/"+(std::string(args[i]).substr(6));
			arg2=true;
		}
		else if(std::string(args[i]).substr(0,3) == "-b="){
			nBlock = std::stoi(std::string(args[i]).substr(3));
		}
		else if(std::string(args[i]).substr(0,3) == "-t="){
			nThread = std::stoi(std::string(args[i]).substr(3));
		}						
	}
	if(!arg1 || !arg2){
		cout<<"Gli argomenti di input sono errati"<<endl;
		cout<<"Inserisci -K=n -file='m', con n = num di sol. max e m = file.txt dei vincoli"<<endl;
		return 0;
	}
	
	string s = firstLine(nomeFile);   //read first line of constraint file
	string infoFirstLine[4];
	stringstream ss(s);
	string word;
	int i = 0;
	while (ss >> word)     //save information
	{
		infoFirstLine[i] = word;
		i++;
	}
	
	int nLitt = stoi(infoFirstLine[2]);             //number of literals
	int nConstr = stoi(infoFirstLine[3]);           //number of constraints
	int nNegPosLit = (nLitt * 2);                  //number of literals (negative and positive)
	long int sizeAdj = nNegPosLit * nNegPosLit;     //size of adj matrix
	bool* adj_matrix = new bool[sizeAdj];             //adj_matrix of all 0
	memset(adj_matrix, false, sizeAdj * sizeof(bool)); 
	
	string* str= new string[nConstr + 1];
	
	funcRead(str, nomeFile);
	
	gpuErrchk(hipGetLastError());
	bool* littExist = new bool[nNegPosLit];   //array of presence in the constraints
	memset(littExist, false, nNegPosLit * sizeof(bool)); 
	
	// #pragma omp parallel shared(str, adj)
	// {
	//  #pragma omp for schedule(auto)
	
	for (int i = 1; i <= nConstr; i++)              //marks the existence of a litteral inside the constriants
	{
		stringstream ss(str[i]);
		string word;
		int pos = 0;
		int pos1 = 0;
		int j = 0;
		while (ss >> word && j <= 1)
		{
			if (j == 0)
			{
				pos = tras(stoi(word), nLitt);
			}
			else if (j == 1)
			{
				pos1 = tras(stoi(word), nLitt);
				adj_matrix[((pos * nNegPosLit) + pos1)] = 1;    //save the existence of constraints between two litterals 
				adj_matrix[((pos1 * nNegPosLit) + pos)] = 1;
				littExist[pos] = true;
				littExist[pos1] = true;
			}
			j++;
		}
	}
	//}
	
	
	bool *d_littExist;                                                          //device litteral existance
	hipMalloc(&d_littExist, nNegPosLit * sizeof(bool));
	hipMemcpy(d_littExist, littExist, nNegPosLit * sizeof(bool), hipMemcpyHostToDevice);
	
	bool *d_adj_matrix;                                                         //device adj matrix
	hipMalloc(&d_adj_matrix, sizeAdj * sizeof(bool));
	hipMemcpy(d_adj_matrix, adj_matrix, sizeAdj * sizeof(bool), hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	
	int status[3] = {0};                                                        //array for status: 0 for add to adj; 1 for -1 and 1 to the same litteral, 2 for the check of similar solution
	int *d_status;
	hipMalloc(&d_status, 3 * sizeof(int));
	
	int* sol=new int[nNegPosLit];                                  //array of current solution
	memset(sol, 0, nNegPosLit * sizeof(int)); 
	int *d_sol;
	hipMalloc((void**)&d_sol, nNegPosLit * sizeof(int));      
	
	hipMemcpy(d_sol, sol, nNegPosLit * sizeof(int), hipMemcpyHostToDevice);     //TODO non so se serve
	
	hipDeviceSynchronize();
	
	hipMemcpy(sol, d_sol, nNegPosLit * sizeof(int), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	
	int* alternativeSol = new int[nNegPosLit];                       //alternative solution to the current solution
	memset(alternativeSol, 0, nNegPosLit * sizeof(int)); 
	int *d_alternativeSol;
	hipMalloc((void**)&d_alternativeSol, nNegPosLit * sizeof(int));
	
	int indexSol = 0;                                           //index of solutions in the solution's array   
	int cSol = 0;                                               //counter of solution
	list<double> prox[1];                                       //TODO
	
	int number = nLitt+1;
	int* solReg= (int*) malloc(nNegPosLit * number*sizeof(int));                            //array of next solutions to check
	if(solReg==NULL)
	printf("ERRORE MEMORIA");
	memset(solReg, 0, nNegPosLit * number * sizeof(int)); 
	
	int *d_solReg;
	hipMalloc((void**)&d_solReg, (nNegPosLit * number) * sizeof(int));
	gpuErrchk(hipMemcpy(d_solReg, solReg, (nNegPosLit * number) * sizeof(int), hipMemcpyHostToDevice));
	
	hipDeviceSynchronize();
	
	int* finalSol=new int[nNegPosLit * k];                              //solution's array   
	int *d_finalSol;
	hipMalloc((void**)&d_finalSol, (nNegPosLit * k) * sizeof(int));
	hipMemcpy(d_finalSol, finalSol, (nNegPosLit * k) * sizeof(int), hipMemcpyHostToDevice);
	
	bool* alreadyVisited=new bool[nNegPosLit];                     //array to save if a node on -1 has already been checked
	memset(alreadyVisited, false, nNegPosLit * sizeof(bool)); 
	
	
	bool *d_alreadyVisited;
	hipMalloc((void**)&d_alreadyVisited, nNegPosLit * sizeof(bool));
	hipMemcpy(d_alreadyVisited, alreadyVisited, nNegPosLit * sizeof(bool), hipMemcpyHostToDevice);
	
	
	auto start = std::chrono::steady_clock::now();
	//creo nuovi archi
	memset(status, 0, 3 * sizeof(int));                                                               //check modifiche
	hipMemcpy(d_status, status, 3 * sizeof(int), hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	createConstraints<<<nBlock, nThread>>>(d_adj_matrix, nNegPosLit, sizeAdj);           //check for same new constraints and for new edge
	hipDeviceSynchronize();
	
	checkDiagonal<<<nBlock, nThread>>>(d_adj_matrix, nNegPosLit, d_sol, d_status);                    //check if the pair of positive and negative is present. If 1 1 and -1 -1 is present, there isn't solution
	hipDeviceSynchronize();
	hipMemcpy(status, d_status, 3 * sizeof(int), hipMemcpyDeviceToHost);
	if(status[0] == 0){
		auto end = std::chrono::steady_clock::now();
		std::chrono::duration<double> elapsed_seconds = end-start;
		cout << "elapsed time: " << elapsed_seconds.count() << "s\n"; 
		
		gpuErrchk(hipGetLastError());
		bool resumeSolution = false; //forse non serve
		i = 0;
		bool esiste = false;
		bool continua = false;
		auto start2 = std::chrono::steady_clock::now();
		
		do{
			continua = false;
			hipMemcpy(d_sol, sol, nNegPosLit * sizeof(int), hipMemcpyHostToDevice);
			hipDeviceSynchronize();
			do{
				memset(status, 0, 3 * sizeof(int));                                               //set status to 0
				hipMemcpy(d_status, status, 3 * sizeof(int), hipMemcpyHostToDevice);            //copy to gpu
				
				if(sol[i] == 0 && sol[i+nLitt] == 0 && resumeSolution == false){                  //start, if two sibling literals both have 0 i.e. no value.
					memcpy(alternativeSol, sol, nNegPosLit*sizeof(int));                            //copy current solution
					
					if(littExist[i]){                                                               //give -1 to solution and 1 to alternative solution
						sol[i] = -1;
						alternativeSol[i] = 1;
						esiste = true;
					}
					if(littExist[i + nLitt]){                                                       //give 1 to solution and -1 to alternative solution
						sol[i + nLitt] = 1;
						alternativeSol[i + nLitt] = -1;
						esiste = true;
					}
					
					if(esiste){
						prox[0].push_back(i);
						gpuErrchk(hipMemcpy(d_alternativeSol, alternativeSol, nNegPosLit * sizeof(int), hipMemcpyHostToDevice));            
						
						gpuErrchk(hipGetLastError());
						saveNextSol<<<nBlock, nThread>>>(d_solReg, d_alternativeSol, nNegPosLit, cSol);                             //save alternative solution
						hipDeviceSynchronize();
						gpuErrchk(hipGetLastError());
						cSol++;
						//printf("\n__%d__\n",cSol);
						gpuErrchk(hipMemcpy(d_sol, sol, nNegPosLit * sizeof(int), hipMemcpyHostToDevice));                                   //copy solution to gpu
						hipDeviceSynchronize();
						resumeSolution = true;   
						esiste = false;
					}
				}
				if(!resumeSolution){
					i++;
				}
				if(resumeSolution){
					checkRow<<<nBlock, nThread>>>(d_adj_matrix, d_sol, nNegPosLit, d_status, d_alreadyVisited, d_littExist);             //insert all 1 to litteral connected to litteral with -1
					hipDeviceSynchronize();
					completeSol<<<nBlock, nThread>>>(d_sol, nNegPosLit, d_littExist);                                                    //if a literal has 1 and its sibling 0, I do -1 and vice versa. In order to complete the solution
					hipDeviceSynchronize();
					hipMemcpy(status, d_status, 3 * sizeof(int), hipMemcpyDeviceToHost);
					hipDeviceSynchronize();
					gpuErrchk(hipMemcpy(sol, d_sol, nNegPosLit * sizeof(int), hipMemcpyDeviceToHost));  //Da migliorare
					hipDeviceSynchronize();
					if(status[0] == 0)                                                                                            //check status of solution completing 
					resumeSolution = false;
					if(status[1]==1){                                                                                                         //check if there is some conflict
						break;
					}
				}
			}while(i < nLitt); 
			
			if(status[1] == 0){                                                                                               //if there is no conflict
				gpuErrchk(hipMemcpy(sol, d_sol, nNegPosLit * sizeof(int), hipMemcpyDeviceToHost));                                       //TODO non serve perchè viene fatto prima????
				hipDeviceSynchronize();
				if(indexSol > 0){                                               
					checkNewSol<<<nBlock, nThread>>>(d_sol, d_finalSol, nNegPosLit, indexSol, d_status);                                         //TODO non serve perchè viene fatto prima????
					hipDeviceSynchronize();                        //check if the found solution already exists 
				}
				if(status[2] == 0 || indexSol == 0){
					k--;
					for (int ssif = 0; ssif < nNegPosLit; ssif++)
					{   
						finalSol[indexSol * nNegPosLit + ssif] = sol[ssif];                                                       //save new solution
					}
					//printf("\n::%d::\n",indexSol * nNegPosLit);
					indexSol++;                                                                                                   //index of solution
					hipMemcpy(d_finalSol, finalSol, (nNegPosLit * k) * sizeof(int), hipMemcpyHostToDevice);
					hipDeviceSynchronize();
				}
			}
			
			memset(status, 0, 3 * sizeof(int));
			hipMemcpy(d_status, status, 3 * sizeof(int), hipMemcpyHostToDevice);
			if (cSol > 0){
				memset(sol, 0, nNegPosLit * sizeof(int));                                                                       //Set status to 0
				i = prox[0].back();                                                                                             //I take position i from which the solution must keep going
				prox[0].pop_back();
				cSol--;
				hipMemcpy(solReg, d_solReg, nNegPosLit * sizeof(int) * (cSol+1), hipMemcpyDeviceToHost);                      //copies the length of the array I need
				
				hipDeviceSynchronize();
				copyNextSol<<<nBlock, nThread>>>(d_solReg, d_sol, nNegPosLit, cSol);                                            //retrieves the last solution that was saved from the solutions to check.
				hipDeviceSynchronize();
				hipMemcpy(sol, d_sol, nNegPosLit * sizeof(int), hipMemcpyDeviceToHost);
				//hipMemcpy(d_alreadyVisited, alreadyVisited, nNegPosLit * sizeof(bool), hipMemcpyHostToDevice);
				resumeSolution = true;
				continua = true;                                                                                                //there are another solution to check
			}
			memset(alreadyVisited, 0, nNegPosLit * sizeof(bool));
			hipMemcpy(d_alreadyVisited, alreadyVisited, nNegPosLit * sizeof(bool), hipMemcpyHostToDevice);
			
		}while (continua && k > 0);     
		
		auto end2 = std::chrono::steady_clock::now();
		std::chrono::duration<double> elapsed_seconds2 = end2-start2;
		cout << "elapsed time: " << elapsed_seconds2.count() << "s\n"; 
		//k,nLitt,nConstr,nBLock,nThread,time,timS
		ofstream myfileD;
		myfileD.open("duration.txt", std::ios_base::app);                                                                                     //save duration in duration.txt
		myfileD  <<indexSol<<";"<<nLitt<<";"<<nConstr<<";"<<nBlock<<";"<<nThread<<";"<< elapsed_seconds.count()<<";"<<elapsed_seconds2.count() <<"s\n";
		myfileD.close();                                                                                   //break the do-while when k = 0 or I have already check all possible solution
		
		ofstream myfile;
		myfile.open ("soluzioni/sol"+nomeFile.substr(8));                                                                                             //save solution in solution.txt
		for(int ind = 0; ind < nNegPosLit*indexSol; ind++){
			myfile << finalSol[ind]<<" ";
			if(ind%nNegPosLit == (nNegPosLit-1) && ind != 0 && ind != (nNegPosLit*indexSol-1))
			myfile << "\n";
		}
		myfile.close();
		cout<<"TERMINATO e k vale ora: "<<k<<" . "; if(k == 0) cout<<"Ci sono tutte le soluzioni che cercavi"<<endl;
	}else{
		cout<<"Conflitto tra i vincoli, non ci sono soluzioni"<<endl;
	}
	hipFree(d_adj_matrix);
	hipFree(d_littExist);
	hipFree(d_status);
	hipFree(d_sol);
	hipFree(d_alternativeSol);
	hipFree(d_finalSol);
	hipFree(d_solReg);
	hipFree(d_alreadyVisited);
	return 0;
}