#include "device.cu"
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <fstream>
#include <bits/stdc++.h>
#include <numeric>
#include <hip/hip_runtime.h>
#include <omp.h>
#include <malloc.h>
#include <unistd.h>
#include <list>


int main(void)  //main
{
  string s = firstLine();   //read first line of constraint file
  string infoFirstLine[4];
  stringstream ss(s);
  string word;
  int i = 0;
  while (ss >> word)     //save information
  {
    infoFirstLine[i] = word;
    i++;
  }

  int nLitt = stoi(infoFirstLine[2]);             //number of literals
  int nConstr = stoi(infoFirstLine[3]);           //number of constraints
  int nNegPosLit = (nLitt * 2);                  //number of literals (negative and positive)
  long int sizeAdj = nNegPosLit * nNegPosLit;     //size of adj matrix
  bool adj_matrix[sizeAdj] = {0};             //adj_matrix of all 0
  string str[nConstr + 1];
  funcRead(str);
  
  bool littExist[nNegPosLit] = {false};   //array of presence in the constraints
  // #pragma omp parallel shared(str, adj)
  // {
  //  #pragma omp for schedule(auto)

  for (int i = 1; i <= nConstr; i++)              //marks the existence of a litteral inside the constriants
  {
    stringstream ss(str[i]);
    string word;
    int pos = 0;
    int pos1 = 0;
    int j = 0;
    while (ss >> word && j <= 1)
    {
      if (j == 0)
      {
        pos = tras(stoi(word), nLitt);
      }
      else if (j == 1)
      {
        pos1 = tras(stoi(word), nLitt);
        adj_matrix[((pos * nNegPosLit) + pos1)] = 1;    //save the existence of constraints between two litterals 
        adj_matrix[((pos1 * nNegPosLit) + pos)] = 1;
      }
      littExist[pos] = true;
      littExist[pos1] = true;
      j++;
    }
  }
  //}


  bool *d_littExist;                                                          //device litteral existance
  hipMalloc(&d_littExist, nNegPosLit * sizeof(bool));
  hipMemcpy(d_littExist, littExist, nNegPosLit * sizeof(bool), hipMemcpyHostToDevice);

  bool *d_adj_matrix;                                                         //device adj matrix
  hipMalloc(&d_adj_matrix, sizeAdj * sizeof(bool));
  hipMemcpy(d_adj_matrix, adj_matrix, sizeAdj * sizeof(bool), hipMemcpyHostToDevice);
  hipDeviceSynchronize();

  int status[3] = {0};                                                        //array for status: 0 for add to adj; 1 for -1 and 1 to the same litteral, 2 for the check of similar solution
  int *d_status;
  hipMalloc(&d_status, 3 * sizeof(int));

  //creo nuovi archi
  createConstraints<<<40, 1024>>>(d_adj_matrix, nNegPosLit, sizeAdj, d_status);           //check for same new constraints and for new edge
  hipDeviceSynchronize();
  hipMemcpy(status, d_status, 3 * sizeof(int), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  cout<<"Bro "<<status[0]<<endl;
  status[0] = 0;
  hipMemcpy(d_status, status, 3 * sizeof(int), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  createConstraints<<<40, 1024>>>(d_adj_matrix, nNegPosLit, sizeAdj, d_status);
  hipDeviceSynchronize();
  hipMemcpy(status, d_status, 3 * sizeof(int), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  cout<<"Bro "<<status[0]<<endl;
  hipDeviceSynchronize();
  //check modifiche
  status[0] = 0;
  hipMemcpy(d_status, status, 3 * sizeof(int), hipMemcpyHostToDevice);
  hipDeviceSynchronize();   
  //check if the pair of positive and negative is present. If 1 1 and -1 -1 is present, there isn't solution
  checkDiagonal<<<40, 1024>>>(d_adj_matrix, nNegPosLit);         
  hipDeviceSynchronize();


  int sol[nNegPosLit] = {0};                                  //array of current solution
  int *d_sol;
  hipMalloc(&d_sol, nNegPosLit * sizeof(int));      
  hipMemcpy(d_sol, sol, nNegPosLit * sizeof(int), hipMemcpyHostToDevice);     //TODO non so se serve

  int alternativeSol[nNegPosLit] = {0};                       //alternative solution to the current solution
  int *d_alternativeSol;
  hipMalloc(&d_alternativeSol, nNegPosLit * sizeof(int));

  int k = 400;                                                  //number of solutiont o find
  int indexSol = 0;                                           //index of solutions in the solution's array   
  int cSol = 0;                                               //counter of solution
  list<double> prox[1];                                       //TODO
  
  int number = nLitt+1;
  int solReg[nNegPosLit * number];                            //array of next solutions to check
  int *d_solReg;
  hipMalloc(&d_solReg, (nNegPosLit * number) * sizeof(int));
  hipMemcpy(d_solReg, solReg, (nNegPosLit * number) * sizeof(int), hipMemcpyHostToDevice);

  
  int finalSol[nNegPosLit * k];                              //solution's array   
  int *d_finalSol;
  hipMalloc(&d_finalSol, (nNegPosLit * k) * sizeof(int));
  hipMemcpy(d_finalSol, finalSol, (nNegPosLit * k) * sizeof(int), hipMemcpyHostToDevice);

  bool alreadyVisited[nNegPosLit] = {0};                     //array to save if a node on -1 has already been checked
  bool *d_alreadyVisited;
  hipMalloc(&d_alreadyVisited, nNegPosLit * sizeof(bool));
  hipMemcpy(d_alreadyVisited, alreadyVisited, nNegPosLit * sizeof(bool), hipMemcpyHostToDevice);
  
  bool resumeSolution = false; //forse non serve
  i = 0;
  bool esiste = false;
  bool continua = false;

  do{
    continua = false;
    hipMemcpy(d_sol, sol, nNegPosLit * sizeof(int), hipMemcpyHostToDevice);
    do{
      memset(status, 0, 3 * sizeof(int));                                               //set status to 0
      hipMemcpy(d_status, status, 3 * sizeof(int), hipMemcpyHostToDevice);            //copy to gpu
      if(sol[i] == 0 && sol[i+nLitt] == 0 && resumeSolution == false){                  //start, if two sibling literals both have 0 i.e. no value.
        memcpy(alternativeSol, sol, nNegPosLit*sizeof(int));                            //copy current solution
        if(littExist[i]){                                                               //give -1 to solution and 1 to alternative solution
          sol[i] = -1;
          alternativeSol[i] = 1;
          esiste = true;
        }
        if(littExist[i + nLitt]){                                                       //give 1 to solution and -1 to alternative solution
          sol[i + nLitt] = 1;
          alternativeSol[i + nLitt] = -1;
          esiste = true;
        }

        if(esiste){
          prox[0].push_back(i);
          hipMemcpy(d_alternativeSol, alternativeSol, nNegPosLit * sizeof(int), hipMemcpyHostToDevice);            
          saveNextSol<<<40, 1024>>>(d_solReg, d_alternativeSol, nNegPosLit, cSol);                             //save alternative solution
          hipDeviceSynchronize();
          cSol++;
          hipMemcpy(d_sol, sol, nNegPosLit * sizeof(int), hipMemcpyHostToDevice);                                   //copy solution to gpu
          resumeSolution = true;   
          esiste = false;
        }
      }
      if(!resumeSolution){
        i++;
      }
      if(resumeSolution){
        checkRow<<<40, 1024>>>(d_adj_matrix, d_sol, nNegPosLit, d_status, d_alreadyVisited, d_littExist);             //insert all 1 to litteral connected to litteral with -1
        hipDeviceSynchronize();
        completeSol<<<40, 1024>>>(d_sol, nNegPosLit, d_littExist);                                                    //if a literal has 1 and its sibling 0, I do -1 and vice versa. In order to complete the solution
        hipDeviceSynchronize();
        hipMemcpy(status, d_status, 3 * sizeof(int), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        hipMemcpy(sol, d_sol, nNegPosLit * sizeof(int), hipMemcpyDeviceToHost);  //Da migliorare
        hipDeviceSynchronize();
        if(status[0] == 0)                                                                                            //check status of solution completing 
          resumeSolution = false;
        if(status[1]==1){                                                                                             //check if there is some conflict
          break;
        }
      }
    }while(i < nLitt); 

    if(status[1] == 0){                                                                                               //if there is no conflict
      hipMemcpy(sol, d_sol, nNegPosLit * sizeof(int), hipMemcpyDeviceToHost);                                       //TODO non serve perchè viene fatto prima????

      if(indexSol > 0)                                                      
        checkNewSol<<<40, 1024>>>(d_sol, d_finalSol, nNegPosLit, indexSol, d_status);                           //check if the found solution already exists 
      if(status[2] == 0 || indexSol == 0){
        k--;
        for (int ssif = 0; ssif < nNegPosLit; ssif++)
        {   
            finalSol[indexSol * nNegPosLit + ssif] = sol[ssif];                                                       //save new solution
        }
        indexSol++;                                                                                                   //index of solution
        hipMemcpy(d_finalSol, finalSol, (nNegPosLit * k) * sizeof(int), hipMemcpyHostToDevice);
      }
    }

    memset(status, 0, 3 * sizeof(int));
    hipMemcpy(d_status, status, 3 * sizeof(int), hipMemcpyHostToDevice);
    if (cSol > 0){
      memset(sol, 0, nNegPosLit * sizeof(int));                                                                       //Set status to 0
      i = prox[0].back();                                                                                             //I take position i from which the solution must keep going
      prox[0].pop_back();
      cSol--;
      hipMemcpy(solReg, d_solReg, nNegPosLit * sizeof(int) * (cSol+1), hipMemcpyDeviceToHost);                      //copies the length of the array I need
      hipDeviceSynchronize();
      
      /*ofstream myfile;
      myfile.open ("prossime.txt");
      for(int ab = 0; ab < nNegPosLit * (cSol+1); ab++){
        if(solReg[ab] == 1 || solReg[ab] == 0){
          myfile << solReg[ab] << "  ";
        }else{
          myfile << solReg[ab] << " ";
        }
        if(ab != 0 && ab%nNegPosLit == 0)
          myfile << "\n";
      }*/
      copyNextSol<<<40, 1024>>>(d_solReg, d_sol, nNegPosLit, cSol);                                            //retrieves the last solution that was saved from the solutions to check.
      hipDeviceSynchronize();
      hipMemcpy(sol, d_sol, nNegPosLit * sizeof(int), hipMemcpyDeviceToHost);
      hipMemcpy(d_alreadyVisited, alreadyVisited, nNegPosLit * sizeof(bool), hipMemcpyHostToDevice);
      resumeSolution = true;
      continua = true;                                                                                                //there are another solution to check
    }

  }while (continua && k > 0);                                                                                         //break the do-while when k = 0 or I have already check all possible solution

  ofstream myfile;
  myfile.open ("solution.txt");                                                                                       //save solution in solution.txt
  for(int ind = 0; ind < nNegPosLit*indexSol; ind++){
    myfile << finalSol[ind]<<" ";
    if(ind%nNegPosLit == (nNegPosLit-1) && ind != 0 && ind != (nNegPosLit*indexSol-1))
      myfile << "\n";
  }
  myfile.close();

  cout<<"TERMINATO e k vale ora: "<<k<<" . "; if(k == 0) cout<<"Ci sono tutte le soluzioni che cercavi"<<endl;
  hipFree(d_adj_matrix);
  hipFree(d_littExist);
  hipFree(d_status);
  hipFree(d_sol);
  hipFree(d_alternativeSol);
  hipFree(d_finalSol);
  hipFree(d_solReg);
  hipFree(d_alreadyVisited);
  return 0;
}