#include <stdio.h>
#include <iostream>
#include <cmath>
#include <fstream>
#include <sstream>
#include <numeric>
#include <hip/hip_runtime.h>
#include <omp.h>
#include <malloc.h>
#include <list>
using namespace std;

void funcRead(string str[], string nomeFile)
{
	string myText;
	int p = 0;
	ifstream MyReadFile(nomeFile);
	while (getline(MyReadFile, myText))
	{
		str[p] = myText;
		p++;
	}

	MyReadFile.close();
}

string firstLine(string nomeFile)
{
	ifstream infile(nomeFile);
	string sLine;
	if (infile.good())
	{
		getline(infile, sLine);
	}

	return sLine;
}

__global__ void createConstraints(bool *d_adj_matrix, int nNegPosLit, long int sizeAdj)
{
	int thid2 = blockIdx.x *blockDim.x + threadIdx.x;
	int thid = 0;
	for (int cont = 0; cont < ceilf(sizeAdj / (gridDim.x *blockDim.x)) + 1; cont++)
	{
		thid = thid2 + cont *(gridDim.x *blockDim.x);

		if (thid < sizeAdj)
		{
			if (d_adj_matrix[thid])
			{
				int secondo = (thid % nNegPosLit);
				if (secondo >= (nNegPosLit / 2))
					secondo = secondo - (nNegPosLit / 2);
				else
					secondo = secondo + (nNegPosLit / 2);
				int primo = floorf(thid / nNegPosLit);
				for (int i = (secondo *nNegPosLit); i < ((secondo + 1) *nNegPosLit); i++)
				{
					int pos = (primo *nNegPosLit) + (i % nNegPosLit);
					if (d_adj_matrix[pos] != 1)
					{
						if (d_adj_matrix[i] && ((i % nNegPosLit) + 1))
						{
							d_adj_matrix[pos] = 1;
							int a = (pos % nNegPosLit) *nNegPosLit;
							int b = floorf(pos / nNegPosLit);
							d_adj_matrix[a + b] = 1; 
						}
					}
				}
			}
		}
	}

	__syncthreads();
}

__global__ void checkDiagonal(bool *adj_matrix, int nNegPosLit, int *d_sol, int *d_status)
{
	int thid2 = blockIdx.x *blockDim.x + threadIdx.x;
	int thid = 0;
	for (int cont = 0; cont < ceilf(nNegPosLit / (blockDim.x *gridDim.x)) + 1; cont++)
	{
		thid = thid2 + cont *(gridDim.x *blockDim.x);
		int thidCheck2 = 0;
		if (thid % (nNegPosLit + 1) == 0 && thid < (nNegPosLit *nNegPosLit))
		{
			thidCheck2 = (nNegPosLit + 1) *(nNegPosLit / 2) + thid;
			if (adj_matrix[thid] == 1 && adj_matrix[thid] == adj_matrix[thidCheck2])
			{
				d_status[0] = d_status[0] || 1;
			}

			if (adj_matrix[thid] == 1 && adj_matrix[thidCheck2] == 0)
			{
				d_sol[thid % nNegPosLit] = 1;
			}
			else if (adj_matrix[thid] == 0 && adj_matrix[thidCheck2] == 1)
			{
				d_sol[thidCheck2 % nNegPosLit] = 1;
			}
		}
	}

	__syncthreads();
}

__global__ void checkRow(bool *d_adj_matrix, int *d_sol, int nNegPosLit, int *d_status, bool *d_alreadyVisited, bool *d_littExist)
{
	int thid2 = blockIdx.x *blockDim.x + threadIdx.x;
	int thid = 0;
	for (int cont = 0; cont < ceilf(nNegPosLit / (blockDim.x *gridDim.x)) + 1; cont++)
	{
		thid = thid2 + cont *(gridDim.x *blockDim.x);
		if (thid < nNegPosLit)
		{
			if (d_sol[thid] == -1 && d_alreadyVisited[thid] == 0)
			{
				for (int i = 0; i < nNegPosLit; i++)
				{
					if (d_adj_matrix[thid *nNegPosLit + i] == 1)
					{
						if (d_sol[i] == 0)
						{
							d_sol[i] = 1;
							if (i >= (nNegPosLit / 2))
							{
								if (d_sol[i - (nNegPosLit / 2)] == 1)
								{
									d_status[1] = d_status[1] || 1;
								}
							}
							else if (i < (nNegPosLit / 2))
							{
								if (d_sol[i + (nNegPosLit / 2)] == 1)
								{
									d_status[1] = d_status[1] || 1;
								}
							}

							d_status[0] = d_status[0] || 1;
						}

						if (d_sol[i] == -1)
						{
							d_status[1] = d_status[1] || 1;
						}

					}
				}

				d_alreadyVisited[thid] = 1;
				if (thid < (nNegPosLit / 2))
					d_alreadyVisited[thid + (nNegPosLit / 2)] = 1;
			}
		}
	}

	__syncthreads();
}

int tras(int number, int
	let)
{
	if (number < 0)
	{
		number = abs(number) +
			let;
	}

	return number - 1;
}

__global__ void completeSol(int *d_sol, int nNegPosLit, bool *d_littExist)
{
	int thid2 = blockIdx.x *blockDim.x + threadIdx.x;
	int thid = 0;
	for (int cont = 0; cont < ceilf(nNegPosLit / (blockDim.x *gridDim.x)) + 1; cont++)
	{
		thid = thid2 + cont *(gridDim.x *blockDim.x);
		if (thid < (nNegPosLit / 2))
		{
			if (d_sol[thid] == 0 && d_sol[thid + (nNegPosLit / 2)] != 0 && d_littExist[thid] == 1)
			{
				if (d_sol[thid + (nNegPosLit / 2)] == 1)
					d_sol[thid] = -1;
				else if (d_sol[thid + (nNegPosLit / 2)] == -1)
					d_sol[thid] = 1;
			}

			if (d_sol[thid] != 0 && d_sol[thid + (nNegPosLit / 2)] == 0 && d_littExist[thid + (nNegPosLit / 2)] == 1)
			{
				if (d_sol[thid] == 1)
					d_sol[thid + (nNegPosLit / 2)] = -1;
				else if (d_sol[thid] == -1)
					d_sol[thid + (nNegPosLit / 2)] = 1;
			}
		}
	}

	__syncthreads();
}

__global__ void saveNextSol(int *d_solReg, int *d_sol_backup, int nNegPosLit, int cSol)
{
	int thid2 = blockIdx.x *blockDim.x + threadIdx.x;
	int thid = 0;
	for (int cont = 0; cont < ceilf(nNegPosLit / (blockDim.x *gridDim.x)) + 1; cont++)
	{
		thid = thid2 + cont *(gridDim.x *blockDim.x);
		if (thid < nNegPosLit)
		{
			d_solReg[(cSol *nNegPosLit) + thid] = d_sol_backup[thid];
		}
	}

	__syncthreads();
}

__global__ void copyNextSol(int *d_solReg, int *d_sol, int nNegPosLit, int cSol)
{
	int thid2 = blockIdx.x *blockDim.x + threadIdx.x;
	int thid = 0;
	for (int cont = 0; cont < ceilf(nNegPosLit / (blockDim.x *gridDim.x)) + 1; cont++)
	{
		thid = thid2 + cont *(gridDim.x *blockDim.x);
		if (thid < nNegPosLit)
		{
			d_sol[thid] = d_solReg[(cSol *nNegPosLit) + thid];
		}
	}

	__syncthreads();
}

__global__ void checkNewSol(int *d_sol, int *d_solFinali, int nNegPosLit, int indexSol, int *d_status)
{
	int thid2 = blockIdx.x *blockDim.x + threadIdx.x;
	int thid = 0;
	for (int cont = 0; cont < ceilf(nNegPosLit / (blockDim.x *gridDim.x)) + 1; cont++)
	{
		thid = thid2 + cont *(gridDim.x *blockDim.x);
		if (thid < indexSol)
		{
			int i = 0;
			while (i < nNegPosLit && d_solFinali[thid *nNegPosLit + i] == d_sol[i])
			{
				i++;
			}

			if (i == nNegPosLit)
				d_status[2] = d_status[2] || 1;
		}
	}

	__syncthreads();
}