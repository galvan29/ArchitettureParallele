#include <stdio.h>
#include <iostream>
#include <cmath>
#include <fstream>
#include <sstream>
#include <numeric>
#include <hip/hip_runtime.h>
#include <omp.h>
#include <malloc.h>
#include <list>


using namespace std;

//gestione e cattura errori GPU
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

void funcRead(string str[], string nomeFile)
{
    string myText;
    int p = 0;
    ifstream MyReadFile(nomeFile);
    while (getline(MyReadFile, myText))
    {
        str[p] = myText;
        p++;
    }
    MyReadFile.close();
}

string firstLine(string nomeFile)
{
    ifstream infile(nomeFile);
    string sLine;
    if (infile.good())
    {
        getline(infile, sLine);
    }
    return sLine;
}

__global__ void createConstraints(bool* d_adj_matrix, int nNegPosLit, long int sizeAdj, int* d_status)
{
    int thid2 = blockIdx.x * blockDim.x + threadIdx.x;
    int thid = 0;
    for (int cont = 0; cont < ceilf((sizeAdj / (blockDim.x * gridDim.x))) + 1; cont++)
    {
        thid = thid2 + cont * (gridDim.x * blockDim.x);

        if (thid < (sizeAdj))
        {
            if (d_adj_matrix[thid])
            {
                int secondo = (thid % nNegPosLit);
                if (secondo >= (nNegPosLit / 2))
                    secondo = secondo - (nNegPosLit / 2);
                else
                    secondo = secondo + (nNegPosLit / 2);
                int primo = floorf(thid / nNegPosLit);
                for (int i = (secondo * nNegPosLit); i < ((secondo + 1) * nNegPosLit); i++)
                {
                    int pos = (primo * nNegPosLit) + (i % nNegPosLit);
                    if (d_adj_matrix[pos] != 1) {
                        if (d_adj_matrix[i] && ((i % nNegPosLit) + 1))// != (primo + 1))
                        {
                            d_adj_matrix[pos] = 1;
                            int a = (pos % nNegPosLit) * nNegPosLit;
                            int b = floorf(pos / nNegPosLit);
                            d_adj_matrix[a + b] = 1;
                            atomicAdd(&d_status[0], 1.0f);
                        }
                    }
                }
            }
        }
    }
    __syncthreads();
}

__global__ void checkDiagonal(bool* adj_matrix, int nNegPosLit, int* d_sol)
{
    int thid2 = blockIdx.x * blockDim.x + threadIdx.x;
    int thid = 0;
    for (int cont = 0; cont < ceilf((nNegPosLit / (blockDim.x * gridDim.x))) + 1; cont++)
    {
        thid = thid2 + cont * (gridDim.x * blockDim.x);
        int thidCheck2 = 0;
        if (thid == 0)
            printf("Sto controllando la diagonale\n");
        if (thid % (nNegPosLit + 1) == 0 && thid < (nNegPosLit * nNegPosLit))
        {
            thidCheck2 = (nNegPosLit + 1) * (nNegPosLit / 2) + thid;
            if (adj_matrix[thid] == 1 && adj_matrix[thid] == adj_matrix[thidCheck2]) {
                printf("Nella status %d e nella status %d hanno entrambi 1\n", thid, thidCheck2);
                printf("Non ci sono soluzioni per il controllo sulla diagonale\n\n");
            }
            if (adj_matrix[thid] == 1 && adj_matrix[thidCheck2] == 0) {
                d_sol[thid % nNegPosLit] = 1;
            }
            else if (adj_matrix[thid] == 0 && adj_matrix[thidCheck2] == 1) {
                d_sol[thidCheck2 % nNegPosLit] = 1;
            }
        }
    }
    __syncthreads();
}

//aggiunge 1 a tutti quelli che sono collegati al -1. se il -1 è già stato controllato allora non lo controlla più

__global__ void checkRow(bool* d_adj_matrix, int* d_sol, int nNegPosLit, int* d_status, bool* d_alreadyVisited, bool* d_littExist) {

    int thid2 = blockIdx.x * blockDim.x + threadIdx.x;
    int thid = 0;
    for (int cont = 0; cont < ceilf((nNegPosLit / (blockDim.x * gridDim.x))) + 1; cont++)
    {
        thid = thid2 + cont * (gridDim.x * blockDim.x);
        if (thid < nNegPosLit) {
            if (d_sol[thid] == -1 && d_alreadyVisited[thid] == 0) {
                for (int i = 0; i < nNegPosLit; i++) {
                    if (d_adj_matrix[thid * nNegPosLit + i] == 1) {
                        if (d_sol[i] == 0) {
                            d_sol[i] = 1;
                            if (i >= (nNegPosLit / 2)) {
                                if (d_sol[i - (nNegPosLit / 2)] == 1) {
                                    d_status[1] = d_status[1] || 1;
                                }
                            }
                            else if (i < (nNegPosLit / 2)) {
                                if (d_sol[i + (nNegPosLit / 2)] == 1) {
                                    d_status[1] = d_status[1] || 1;
                                }
                            }
                            d_status[0] = d_status[0] || 1;
                        }
                        if (d_sol[i] == -1) {
                            //printf("Questa soluzione non va bene\n");
                            d_status[1] = d_status[1] || 1;
                        }
                        // printf("Trovato\n");
                    }
                }
                d_alreadyVisited[thid] = 1;
                if (thid < (nNegPosLit / 2))
                    d_alreadyVisited[thid + (nNegPosLit / 2)] = 1;
            }
        }
    }
    __syncthreads();
}

int tras(int number, int let)
{
    if (number < 0)
    {
        number = abs(number) + let;
    }
    return number - 1;
}

__global__ void completeSol(int* d_sol, int nNegPosLit, bool* d_littExist)
{
    int thid2 = blockIdx.x * blockDim.x + threadIdx.x;
    int thid = 0;
    for (int cont = 0; cont < ceilf((nNegPosLit / (blockDim.x * gridDim.x))) + 1; cont++)
    {
        thid = thid2 + cont * (gridDim.x * blockDim.x);
        //int thid = 0;
        if (thid < (nNegPosLit / 2))
        {
            if (d_sol[thid] == 0 && d_sol[thid + (nNegPosLit / 2)] != 0 && d_littExist[thid] == 1)
            {
                if (d_sol[thid + (nNegPosLit / 2)] == 1)
                    d_sol[thid] = -1;
                else if (d_sol[thid + (nNegPosLit / 2)] == -1)
                    d_sol[thid] = 1;
            }
            if (d_sol[thid] != 0 && d_sol[thid + (nNegPosLit / 2)] == 0 && d_littExist[thid + (nNegPosLit / 2)] == 1)
            {
                if (d_sol[thid] == 1)
                    d_sol[thid + (nNegPosLit / 2)] = -1;
                else if (d_sol[thid] == -1)
                    d_sol[thid + (nNegPosLit / 2)] = 1;
            }
        }
    }
    __syncthreads();
}

__global__ void saveNextSol(int* d_solReg, int* d_sol_backup, int nNegPosLit, int cSol) {
    int thid2 = blockIdx.x * blockDim.x + threadIdx.x;
    int thid = 0;
    for (int cont = 0; cont < ceilf((nNegPosLit / (blockDim.x * gridDim.x))) + 1; cont++)
    {
        thid = thid2 + cont * (gridDim.x * blockDim.x);
        if (thid < nNegPosLit) {
            //printf("HO COPIATO %d\n" , (cSol * nNegPosLit) + thid);
            d_solReg[(cSol * nNegPosLit) + thid] = d_sol_backup[thid];
        }
    }
    __syncthreads();
}

__global__ void copyNextSol(int* d_solReg, int* d_sol, int nNegPosLit, int cSol) {
    int thid2 = blockIdx.x * blockDim.x + threadIdx.x;
    int thid = 0;
    for (int cont = 0; cont < ceilf((nNegPosLit / (blockDim.x * gridDim.x))) + 1; cont++)
    {
        thid = thid2 + cont * (gridDim.x * blockDim.x);
        if (thid < nNegPosLit) {
            d_sol[thid] = d_solReg[(cSol * nNegPosLit) + thid];
            //printf("HO COPIATO\n");
        }
    }
    __syncthreads();
}

__global__ void checkNewSol(int* d_sol, int* d_solFinali, int nNegPosLit, int indexSol, int* d_status) {         //TODO VERIFICARE
    int thid2 = blockIdx.x * blockDim.x + threadIdx.x;
    int thid = 0;
    for (int cont = 0; cont < ceilf((nNegPosLit / (blockDim.x * gridDim.x))) + 1; cont++)
    {
        thid = thid2 + cont * (gridDim.x * blockDim.x);
        if (thid < indexSol) {
            int i = 0;
            while (i < nNegPosLit && d_solFinali[thid * nNegPosLit + i] == d_sol[i]) {
                i++;
            }

            if (i == nNegPosLit)
                d_status[2] = d_status[2] || 1;
        }
    }
    __syncthreads();
}