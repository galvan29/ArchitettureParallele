#include <stdio.h>
#include <iostream>
#include <cmath>
#include <fstream>
#include <sstream>
#include <numeric>
#include <hip/hip_runtime.h>
#include <omp.h>
#include <malloc.h>
#include <list>


using namespace std;

//gestione e cattura errori GPU
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void funcRead(string str[], string nomeFile)
{
  string myText;
  int p = 0;
  ifstream MyReadFile(nomeFile);
  while (getline(MyReadFile, myText))
  {
    str[p] = myText;
    p++;
  }
  MyReadFile.close();
}

string firstLine(string nomeFile)
{
  ifstream infile(nomeFile);
  string sLine;
  if (infile.good())
  {
    getline(infile, sLine);
  }
  return sLine;
}

__device__ void diagonal(bool *d_adj_matrix, int nNegPosLit, int thid)
{
  int secondo = (thid % nNegPosLit);
  int primo = floorf(thid / nNegPosLit);
  d_adj_matrix[(primo * nNegPosLit) + secondo] |= d_adj_matrix[(secondo * nNegPosLit) + primo];
}

__global__ void createConstraints(bool *d_adj_matrix, int nNegPosLit, long int sizeAdj, int *d_status)
{
  int thid2 = blockIdx.x * blockDim.x + threadIdx.x;
  int thid = 0;
  for (int Pass = 0; Pass < ceilf((sizeAdj / (blockDim.x * gridDim.x))) + 1; Pass++)
  {
    thid = thid2 + Pass * (gridDim.x * blockDim.x);

    if (thid < (sizeAdj))
    {
      if (d_adj_matrix[thid])
      { 
        int secondo = (thid % nNegPosLit);
        if (secondo >= (nNegPosLit / 2))
          secondo = secondo - (nNegPosLit / 2);
        else
          secondo = secondo + (nNegPosLit / 2);                               
        int primo = floorf(thid / nNegPosLit); 
        for (int i = (secondo * nNegPosLit); i < ((secondo + 1) * nNegPosLit); i++)
        { 
          int pos = (primo * nNegPosLit) + (i % nNegPosLit);
          if(d_adj_matrix[pos] != 1){
            if (d_adj_matrix[i] && ((i % nNegPosLit) + 1) != (primo + 1))
            {    
              d_adj_matrix[pos] = 1;
              int a = (pos%nNegPosLit)*nNegPosLit;
              int b = floorf(pos/nNegPosLit);
              d_adj_matrix[a + b] = 1;
              atomicAdd(&d_status[0], 1.0f); 
            } 
          }
        }
      }
    }
    __syncthreads();

    if (thid < (nNegPosLit * nNegPosLit))
    {
      diagonal(d_adj_matrix, nNegPosLit, thid);
    }
  }
  __syncthreads();
}

__global__ void checkDiagonal(bool *adj_matrix, int nNegPosLit)
{
  int thid = blockIdx.x * blockDim.x + threadIdx.x;
  int thidCheck1 = 0;
  int thidCheck2 = 0;
  if (thid == 0)
    printf("Ho controllato la diagonale per la presenza of per esempio 2 2 and -2 -2\n");
  if (thid % (nNegPosLit + 1) == 0 && thid < (nNegPosLit * nNegPosLit))
  {
    thidCheck1 = thid;
    thidCheck2 = (nNegPosLit + 1) * (nNegPosLit / 2) + thid;
    if (adj_matrix[thidCheck1] == 1 && adj_matrix[thidCheck1] == adj_matrix[thidCheck2])
    {
      printf("Nella status %d e nella status %d hanno entrambi 1\n", thidCheck1, thidCheck2);
      printf("Quindi teoricamente non ci sono soluzioni\n\n");
    }
  }
  __syncthreads();
}

//aggiunge 1 a tutti quelli che sono collegati al -1. se il -1 è già stato controllato allora non lo controlla più

__global__ void checkRow(bool *d_adj_matrix, int *d_sol, int nNegPosLit, int *d_status, bool *d_alreadyVisited, bool *d_littExist){
  int thid = blockIdx.x * blockDim.x + threadIdx.x;
  if(thid < nNegPosLit){
    if(d_sol[thid] == -1 && d_alreadyVisited[thid] == 0){
      for(int i = 0; i < nNegPosLit; i++){
        if(d_adj_matrix[thid*nNegPosLit+i] == 1){
          if(d_sol[i] == 0){
            d_sol[i] = 1;
            if(i >= (nNegPosLit/2)){
              if(d_sol[i-(nNegPosLit/2)] == 1){
                d_status[1] = d_status[1] || 1;
              }
            }else if(i < (nNegPosLit/2)){
              if(d_sol[i+(nNegPosLit/2)] == 1){
                d_status[1] = d_status[1] || 1;
              }
            }
            d_status[0] = d_status[0] || 1;
          }
          if(d_sol[i] == -1){
            //printf("Questa soluzione non va bene\n");
            d_status[1] = d_status[1] || 1;
          }
          // printf("Trovato\n");
        }
      }
      d_alreadyVisited[thid] = 1;
      if(thid < (nNegPosLit/2))
        d_alreadyVisited[thid+(nNegPosLit/2)] = 1;
    }
  }
  __syncthreads();
}

int tras(int number, int let)
{
  if (number < 0)
  {
    number = abs(number) + let;
  }
  return number - 1;
}

__global__ void completeSol(int *d_sol, int nNegPosLit, bool *d_littExist)
{
  int thid = blockIdx.x * blockDim.x + threadIdx.x;
  //int thid = 0;
  if (thid < (nNegPosLit / 2))
  {
    if (d_sol[thid] == 0 && d_sol[thid + (nNegPosLit / 2)] != 0 && d_littExist[thid] == 1)
    {
      if (d_sol[thid + (nNegPosLit / 2)] == 1)
        d_sol[thid] = -1;
      else if (d_sol[thid + (nNegPosLit / 2)] == -1)
        d_sol[thid] = 1;
    }
    if (d_sol[thid] != 0 && d_sol[thid + (nNegPosLit / 2)] == 0 && d_littExist[thid + (nNegPosLit / 2)] == 1)
    {
      if (d_sol[thid] == 1)
        d_sol[thid + (nNegPosLit / 2)] = -1;
      else if (d_sol[thid] == -1)
        d_sol[thid + (nNegPosLit / 2)] = 1;
    }
  }
__syncthreads();
}

__global__ void saveNextSol(int *d_solReg, int *d_sol_backup, int nNegPosLit, int cSol){
  int thid = blockIdx.x * blockDim.x + threadIdx.x;
  if(thid < nNegPosLit){
    //printf("HO COPIATO %d\n" , (cSol * nNegPosLit) + thid);
    d_solReg[(cSol * nNegPosLit) + thid] = d_sol_backup[thid];
  }
  __syncthreads();
}

__global__ void copyNextSol(int *d_solReg, int *d_sol, int nNegPosLit, int cSol){
  int thid = blockIdx.x * blockDim.x + threadIdx.x;
  if(thid < nNegPosLit){
    d_sol[thid] = d_solReg[(cSol * nNegPosLit) + thid];
    //printf("HO COPIATO\n");
  }
  __syncthreads();
}

__global__ void checkNewSol(int *d_sol, int *d_solFinali, int nNegPosLit, int indexSol, int *d_status){         //TODO VERIFICARE
  int thid = blockIdx.x * blockDim.x + threadIdx.x;
  if(thid < indexSol){
    int i = 0;
    while(i < nNegPosLit && d_solFinali[thid*nNegPosLit + i] == d_sol[i]){
      i++;
    }

    if(i==nNegPosLit)
      d_status[2] = d_status[2] || 1;
  }
  __syncthreads();
}