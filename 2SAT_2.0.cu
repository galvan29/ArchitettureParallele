#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <fstream>
#include <bits/stdc++.h>
#include <numeric>
#include <hip/hip_runtime.h>
#include <omp.h>
#include <malloc.h>
#include <unistd.h>
#include <list>
#include <bits/stdc++.h>
// Utilizza deque #include <deque>
// Hai efficienza ad aggiungere ai lati 

using namespace std;

void funcRead(string str[])
{
  string myText;
  int p = 0;
  ifstream MyReadFile("vincoli.txt");
  while (getline(MyReadFile, myText))
  {
    str[p] = myText;
    p++;
  }
  MyReadFile.close();
}

string firstLine()
{
  ifstream infile("vincoli.txt");
  string sLine;
  if (infile.good())
  {
    getline(infile, sLine);
  }
  return sLine;
}

__device__ void diagonale(bool *d_matrix, int length, int thid)
{
  int secondo = (thid % length);
  int primo = floorf(thid / length);
  d_matrix[(primo * length) + secondo] |= d_matrix[(secondo * length) + primo];
}

__global__ void prova(bool *d_matrix, int length, long int lengthx2, int *d_posizione)
{
  int thid2 = blockIdx.x * blockDim.x + threadIdx.x;
  int thid = 0;
  /*if(thid2 == 0){
    printf("Ma quante volte rientro\n");
  }*/

  for (int Pass = 0; Pass < ceilf((lengthx2 / (blockDim.x * gridDim.x))) + 1; Pass++)
  {
    thid = thid2 + Pass * (gridDim.x * blockDim.x);

    if (thid < (lengthx2))
    {
      if (d_matrix[thid])
      { // thid = 4
        int secondo = (thid % length);
        if (secondo >= (length / 2))
          secondo = secondo - (length / 2);
        else
          secondo = secondo + (length / 2);                                 // 4
        int primo = floorf(thid / length); // 0
        for (int i = (secondo * length); i < ((secondo + 1) * length); i++)
        { // da 24
          if (d_matrix[i] && ((i % length) + 1) != (primo + 1))
          { // 24 però 24%6+1 == 1 quindi non entro
            int posizione = (primo * length) + (i % length);
            d_matrix[posizione] = 1;
            d_posizione[0] = 1;
          }
        }
      }
      //sistema(d_matrix, d_matrix2, length, thid);   //3 0
      //sistema2(d_matrix, d_matrix2, length, thid);
    }
    __syncthreads();

    if (thid < (length * length))
    {
      diagonale(d_matrix, length, thid);
    }
  }
  __syncthreads();
}

// LA DIAGONALE NON MI TORNA
__global__ void checkDiagonale(bool *matrix, int length)
{
  int thid = blockIdx.x * blockDim.x + threadIdx.x;
  int thidCheck1 = 0;
  int thidCheck2 = 0;
  if (thid % (length + 1) == 0 && thid < (length * length))
  {
    thidCheck1 = thid;
    thidCheck2 = (length + 1) * (length / 2) + thid;
    if (matrix[thidCheck1] == 1 && matrix[thidCheck1] == matrix[thidCheck2])
    {
      printf("Nella posizione %d e nella posizione %d hanno entrambi 1\n", thidCheck1, thidCheck2);
      printf("Quindi teoricamente non ci sono soluzioni\n\n");
    }
  }
  if (thid == 0)
    printf("Ho controllato la diagonale\n");
  __syncthreads();
}


bool checkBoolArray(bool *daVis, int length)
{
  int i = 0;
  while (i < length)
  {
    if (daVis[i])
      return true;
    i++;
  }
  return false;
}

double trasformaDaArrayAIntNeg(int *sol, int length)
{
  double somma = 0;
  for (int i = (length - 1); i >= 0; i--)
  {
    if (sol[i] == -1)
    {
      somma += pow(2, (length - 1) - i);
    }
  }
  cout<<"Valore negativo "<<somma<<endl;
  return somma;
}

double trasformaDaArrayAIntPos(int *sol, int length)
{
  double somma = 0;
  for (int i = (length - 1); i >= 0; i--)
  {
    if (sol[i] == 1)
    {
      somma += pow(2, (length - 1) - i);
    }
  }
  cout<<"Valore positivo "<<somma<<endl;
  return somma;
}

void trasformaDaArrayAArray(int *sol, int length, int *temp)
{
  for (int i = 0; i < length; i++)
  {
    if (sol[i] == 1)
      sol[i] = 1;
    if (temp[i] == 1)
      sol[i] = 1;
  }
}

void trasformaDaIntAArrayPos(int *sol, int length, int val)
{
  for (int i = (length - 1); i >= 0; i--)
  {
    if (val > 0)
    {
      sol[i] = val % 2;
      val = val / 2;
    }
  }
}

void trasformaDaIntAArrayNeg(int *sol, int length, int val)
{
  for (int i = (length - 1); i >= 0; i--)
  {
    if (val > 0)
    {
      sol[i] = val % 2;
      val = val / 2;
      sol[i] = sol[i]*(-1);
    }
  }
}



__global__ void checkRow(bool *d_matrix, int *d_sol, int length, int *d_posizione, bool *d_visitato, bool *d_esisteNeiVincoli){
  int thid = blockIdx.x * blockDim.x + threadIdx.x;
  //int thid = 0;
    //printf("Sono thid %d e ho valore %d stato visitato? %d\n", thid, d_sol[thid], d_visitato[thid]);
  if(thid < length && d_sol[thid] == -1 && d_visitato[thid] == 0){
    for(int i = 0; i < length; i++){
      if(d_matrix[thid*length+i] == 1){
        if(d_sol[i] == 0){
          d_sol[i] = 1;
          if(i >= (length/2)){
            if(d_sol[i-(length/2)] == 1){
              d_posizione[1] = d_posizione[1] || 1;
            }
          }else if(i < (length/2)){
            if(d_sol[i+(length/2)] == 1){
              d_posizione[1] = d_posizione[1] || 1;
            }
          }
          d_posizione[0] = d_posizione[0] || 1;
        }
        if(d_sol[i] == -1){
          //printf("Questa soluzione non va bene\n");
          d_posizione[1] = d_posizione[1] || 1;
        }
        // printf("Trovato\n");
      }
    }
    d_visitato[thid] = 1;
    d_visitato[thid+(length/2)] = 1;
  }
  __syncthreads();
}

#include <memory>
void printBool(bool *array, int length){
  
  for(int i = 0; i < length; i++){
    cout<<array[i]<<" ";
  }
  cout<<endl;
}
void printInt(int *array, int length){
  
  for(int i = 0; i < length; i++){
    cout<<array[i]<<" ";
  }
  cout<<endl;
}


#include <vector>
#include <algorithm>
bool cheK(std::vector<int> &sol){
  return std::any_of(sol.begin(), sol.end(), [](const int &i){return i == 0;});
}


int tras(int number, int let)
{
  if (number < 0)
  {
    number = abs(number) + let;
  }
  return number - 1;
}

bool checkIfSolZero(int *sol, int nTotLet)
{
  for (int i = 0; i < nTotLet; i++)
  {
    if (sol[i] == 0)
    {
      return true;
    }
  }
  return true;
}

__global__ void checkSolution(int *d_sol, int length, int * d_posizione, bool *d_matrix){
  int thid = blockIdx.x * blockDim.x + threadIdx.x;
  if(thid < length){
    //printf("Sono %d\n", thid);
    for(int i = thid*length; i < thid*(1+length); i++){
      if(d_matrix[i] == 1 && d_sol[thid] == -1 && d_sol[i%length] == -1){
        d_posizione[1] = d_posizione[1] || 1;
      }
    } 
  }

}

__global__ void completaSol(int *d_sol, int length, bool *d_esisteNeiVincoli)
{
  int thid = blockIdx.x * blockDim.x + threadIdx.x;
  //int thid = 0;
  if (thid < (length / 2))
  {
    if (d_sol[thid] == 0 && d_sol[thid + (length / 2)] != 0 && d_esisteNeiVincoli[thid] == 1)
    {
      if (d_sol[thid + (length / 2)] == 1)
        d_sol[thid] = -1;
      else if (d_sol[thid + (length / 2)] == -1)
        d_sol[thid] = 1;
    }
    if (d_sol[thid] != 0 && d_sol[thid + (length / 2)] == 0 && d_esisteNeiVincoli[thid + (length / 2)] == 1)
    {
      if (d_sol[thid] == 1)
        d_sol[thid + (length / 2)] = -1;
      else if (d_sol[thid] == -1)
        d_sol[thid + (length / 2)] = 1;
    }
  }
__syncthreads();
}

int main(void)
{
  string s = firstLine();
  string arrayyy[4];
  stringstream ss(s);
  string word;
  int i = 0;
  while (ss >> word)
  {
    arrayyy[i] = word;
    i++;
  }

  int letterali = stoi(arrayyy[2]);
  int vincoli = stoi(arrayyy[3]);
  int nTotLet = (letterali * 2);
  long int nTotLetx2 = nTotLet * nTotLet;
  bool matrix[nTotLetx2] = {0};
  string str[vincoli + 1];
  funcRead(str);
  
  bool esisteNeiVincoli[nTotLet] = {0};
  // #pragma omp parallel shared(str, adj)
  // {
  //  #pragma omp for schedule(auto)
  for (int i = 1; i <= vincoli; i++)
  {
    stringstream ss(str[i]);
    string word;
    int pos = 0;
    int pos1 = 0;
    int j = 0;
    while (ss >> word && j <= 1)
    {
      if (j == 0)
      {
        pos = tras(stoi(word), letterali);
      }
      else if (j == 1)
      {
        pos1 = tras(stoi(word), letterali);
        matrix[((pos * nTotLet) + pos1)] = 1;
        matrix[((pos1 * nTotLet) + pos)] = 1;
      }
      esisteNeiVincoli[pos] = true;
      esisteNeiVincoli[pos1] = true;
      j++;
    }
  }
  bool *d_esisteNeiVincoli;
  hipMalloc(&d_esisteNeiVincoli, nTotLet * sizeof(bool));
  hipMemcpy(d_esisteNeiVincoli, esisteNeiVincoli, nTotLet * sizeof(bool), hipMemcpyHostToDevice);

  /*for(int i=0; i<nTotLet; i++){
    cout<<esisteNeiVincoli[i]<<" ";
    if((i+1) == (nTotLet/2))
      cout<<endl;
  }
  cout<<endl;
  //} */

  // https://docs.nvidia.com/cuda/cusparse/index.html#coo-format

  bool *d_matrix;
 // bool *d_matrix2;
  //bool *d_matrix3;

  hipMalloc(&d_matrix, nTotLetx2 * sizeof(bool));
  //hipMalloc(&d_matrix2, nTotLetx2 * sizeof(bool));
  //hipMalloc(&d_matrix3, nTotLetx2 * sizeof(bool));

  hipMemcpy(d_matrix, matrix, nTotLetx2 * sizeof(bool), hipMemcpyHostToDevice);
 // hipMemcpy(d_matrix3, matrix, nTotLetx2 * sizeof(bool), hipMemcpyHostToDevice);
 // sleep(10);
  // bool out[nTotLet];
  // bool *d_out;
  // hipMalloc(&d_out, nTotLetx2*sizeof(bool));

  hipDeviceSynchronize();

  int posizione[2] = {0};
  int *d_posizione;
  hipMalloc(&d_posizione, 2 * sizeof(int));


  prova<<<40, 1024>>>(d_matrix, nTotLet, nTotLetx2, d_posizione);
  hipDeviceSynchronize();
  prova<<<40, 1024>>>(d_matrix, nTotLet, nTotLetx2, d_posizione);
  hipDeviceSynchronize();

      
  posizione[0] = 0;
  hipMemcpy(d_posizione, posizione, 2 * sizeof(int), hipMemcpyHostToDevice);
          
    

  //hipFree(d_matrix);
  //hipFree(d_matrix2);
  checkDiagonale<<<40, 1024>>>(d_matrix, nTotLet);

  hipDeviceSynchronize();

  // PROVIAMO A CERCARE UNA SOLUZIONE
  int sol[nTotLet] = {0};
  int *d_sol;
  hipMalloc(&d_sol, nTotLet * sizeof(int));
  //hipMemcpy(d_sol, sol, nTotLet * sizeof(int), hipMemcpyHostToDevice);
  int sol_backup[nTotLet] = {0};
  int *d_sol_backup;
  hipMalloc(&d_sol_backup, nTotLet * sizeof(int));

  //bool daVis[nTotLet];
  //bool *d_daVis;
  //hipMalloc(&d_daVis, nTotLet * sizeof(bool));
  // hipMemcpy(d_daVis, daVis, nTotLet*sizeof(int), hipMemcpyHostToDevice);
  
  
  
  int k = 70;
  int cSol = 0;
  list<double> prox[100];
  list<bool> soluzioniRegistrate[k];

  bool visitato[nTotLet] = {0};
  bool *d_visitato;

  hipMalloc(&d_visitato, nTotLet * sizeof(bool));
  hipMemcpy(d_visitato, visitato, nTotLet * sizeof(bool), hipMemcpyHostToDevice);
  hipMemcpy(d_sol, sol, nTotLet * sizeof(int), hipMemcpyHostToDevice);
  bool riprendoSoluzione = false;
  int temp[nTotLet];

  i = 0;

  bool esiste = false;
  bool continua = false;
  do{
    continua = false;
    hipMemcpy(d_sol, sol, nTotLet * sizeof(int), hipMemcpyHostToDevice);
    do{
      posizione[0] = 0;
      hipMemcpy(d_posizione, posizione, 2 * sizeof(int), hipMemcpyHostToDevice);
      //cout<<"Soluzione prima di dargli i valori"<<endl;
      //printInt(sol, nTotLet);
      //cout<<endl;
      if(sol[i] == 0 && sol[i+letterali] == 0 && riprendoSoluzione == false){
        memcpy(sol_backup, sol, nTotLet*sizeof(int));
        if(esisteNeiVincoli[i]){
          sol[i] = -1;
          sol_backup[i] = 1;
          esiste = true;
        }
        if(esisteNeiVincoli[i + letterali]){
          sol[i + letterali] = 1;
          sol_backup[i + letterali] = -1;
          esiste = true;
        }
        if(esiste){
         //cout<<endl<<"Provo"<<endl;
         //printInt(sol, nTotLet);
         // cout<<endl;
          prox[0].push_back(trasformaDaArrayAIntNeg(sol_backup, nTotLet));
          prox[0].push_back(i);
          prox[0].push_back(trasformaDaArrayAIntPos(sol_backup, nTotLet));
         cout<<"-------------------------------------------------"<<endl;
         cout<<"Soluzione salvata che continuerò"<<endl;
          cout<<"-------------------------------------------------"<<endl;
          printInt(sol_backup, nTotLet);
          cout<<endl; 
          hipMemcpy(d_sol, sol, nTotLet * sizeof(int), hipMemcpyHostToDevice);
        }
      }
      if(!riprendoSoluzione){
        if(esiste){
          //cout<<"Trovato da dare un nuovo valore"<<endl;
          riprendoSoluzione = true;   
          esiste = false;       
        }
        i++;
        //cout<<"Aumentato i, vale "<<i<<endl;
      }
      if(riprendoSoluzione){
        checkRow<<<40, 1024>>>(d_matrix, d_sol, nTotLet, d_posizione, d_visitato, d_esisteNeiVincoli);
        hipDeviceSynchronize();
        hipMemcpy(posizione, d_posizione, 2 * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(sol, d_sol, nTotLet * sizeof(int), hipMemcpyDeviceToHost);  //Da migliorare
       //cout<<"posizione 0 "<<posizione[0]<<endl;
       //cout<<"posizione 1 "<<posizione[1]<<endl;
        if(posizione[0] == 0)
          riprendoSoluzione = false;
      }
     // cout<<"Valore del controllo errori "<<posizione[1]<<endl;
    }while(i < letterali && posizione[1] == 0);   //trovare modo per smettere prima

    hipMemcpy(sol, d_sol, nTotLet * sizeof(int), hipMemcpyDeviceToHost);   
    //cout<<endl;
    //cout<<"Soluzione:"<<endl;
    //checkSolution<<<40, 1024>>>(d_sol, nTotLet, d_posizione, d_matrix);
    hipDeviceSynchronize();
    hipMemcpy(posizione, d_posizione, 2 * sizeof(int), hipMemcpyDeviceToHost);
    if(posizione[1] == 1){
      for (int ssif = 0; ssif < nTotLet; ssif++)
      {
        cout << sol[ssif] << " ";
      }
      cout<<"Non la salvo"<<endl;
    }
    else if(posizione[1] == 0){
      for (int ssif = 0; ssif < nTotLet; ssif++)
      {
        cout << sol[ssif] << " ";
      }
      cout<<"Trovata una soluzione"<<endl;
      k--;

      // Questo serve? Mi sa di si
      completaSol<<<40, 1024>>>(d_sol, nTotLet, d_esisteNeiVincoli);

      hipMemcpy(sol, d_sol, nTotLet * sizeof(int), hipMemcpyDeviceToHost); 
      cout<<endl<<"Soluzione sistemata"<<endl;
      for (int ssif = 0; ssif < nTotLet; ssif++)
      {
        cout << sol[ssif] << " ";
      }
      cout<<endl;



    }
      //cout<<"La salvo"<<endl;
    //cout<<endl;

    posizione[1] = 0;
    hipMemcpy(d_posizione, posizione, 2 * sizeof(int), hipMemcpyHostToDevice);
    if (prox[0].size() > 0){
      memset(sol, 0, nTotLet * sizeof(int));
      trasformaDaIntAArrayPos(temp, nTotLet, prox[0].back());
      prox[0].pop_back();

      cout<<"Positivo"<<endl;
      printInt(temp, nTotLet);

      i = prox[0].back();
      prox[0].pop_back();

      trasformaDaIntAArrayNeg(sol, nTotLet, prox[0].back());
      prox[0].pop_back();
      cout<<"Negativo"<<endl;
      printInt(sol, nTotLet);
      trasformaDaArrayAArray(sol, nTotLet, temp);
      printInt(sol, nTotLet);      
      hipMemcpy(d_visitato, visitato, nTotLet * sizeof(bool), hipMemcpyHostToDevice);
      cout<<"^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^"<<endl;
      cout<<"Riprendo questa soluzione"<<endl;
      cout<<"^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^"<<endl;
      riprendoSoluzione = true;
      //printInt(sol, nTotLet);
      //cout<<endl;
      continua = true;
    }


  }while (continua && k > 0);
  cout<<endl;
  cout<<"TERMINATO"<<endl;
  cout<<"k vale ora: "<<k<<endl;
  if(k == 0)
    cout<<"Ci sono tutte le soluzioni che cercavi"<<endl;
 /* cout << "Soluzioni mostrate in ordine di registrazione in valore intero: " << endl;
  cout << "Dal decimale al bin rendo 1 gli 1 e i -1 0: " << endl;
  soluzioniRegistrate[0].sort();
  soluzioniRegistrate[0].unique();
  while (soluzioniRegistrate[0].size() > 0)
  {
    cout << soluzioniRegistrate[0].front() << endl;
    soluzioniRegistrate[0].pop_front();
  }
  cout << endl;*/
  hipFree(d_matrix);
  return 0;
}
