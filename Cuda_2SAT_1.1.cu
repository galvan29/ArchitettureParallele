/*

ArrayList<Integer[]> values = new ArrayList<>(); 
values.add(new Integer[] { 2, -4 }); 
int thid = blockIdx.x * blockDim.x + threadIdx.x;
int thidI = 0;
for(int Pass=0; Pass<ceilf((len/(blockDim.x)))+1; Pass++){
thidI = thid + Pass*(gridDim.x*blockDim.x );
if(thidI<len){

if(!CHECK[thidI] && *out){
*out=false;
}
}
}
*/

#include <stdio.h>
#include <iostream>
#include <cmath>
#include <fstream>
#include <bits/stdc++.h>
#include <numeric> 
#include <hip/hip_runtime.h>
#include <omp.h>
using namespace std;

__global__ void trasformaDecBin(int arr[], int val, int mat[], int vincoli, bool *CHECK, int letterali){
}



void print(list<int>& mylist,
  int index)
  {
    cout << "The list elements stored at the index " << 
    index << ": \n";
    
    for (auto element : mylist) 
    {
      cout<<element<<" ";
    }
    cout << '\n';
  }
  
  void print(list<int>* myContainer, int n)
  {
    cout << "adj elements:\n\n";
    for (int i = 1; i < n; i++) 
    {
      print(myContainer[i], i);
    }
  }
  
  
  
  void funcRead(string str[]){
    string myText;
    int p = 0;
    ifstream MyReadFile("vincoli.txt");
    while (getline (MyReadFile, myText)) {
      str[p] = myText;
      p++;
    }
    MyReadFile.close();
  }
  
  string firstLine(){
    ifstream infile("vincoli.txt");
    string sLine;
    if (infile.good()){
      getline(infile, sLine);
    }
    return sLine;
  }
  
  int main(void)
  { 
    string s = firstLine();
    string arrayyy[4];
    stringstream ss(s);
    string word;
    int i=0;
    while (ss >> word) {
      arrayyy[i]=word;
      i++;
    }
    int letterali = stoi(arrayyy[2]);
    int vincoli = stoi(arrayyy[3]);
    int nTotLet = (letterali*2)+1;
    list<int> adj[nTotLet];
    list<int> adj2[nTotLet];
    
    string str[vincoli+1];
    funcRead(str);
    
    //int *letter, *d_letter;
    //letter = (int*)malloc(letterali*sizeof(int));
    
   // #pragma omp parallel shared(str, adj)
   // { 
    //  #pragma omp for schedule(auto)
      for(int i=1; i<(vincoli+1); i++){
        stringstream ss(str[i]);
        string word;
        int j = 0;
        int pos = 0;
        int pos1 = 0;

        while (ss >> word && j<=1) {
          if(j==0)
            pos = stoi(word);
          if(j==1){
            pos1=stoi(word);
            if(pos < 0)
              pos = abs(pos)+letterali;
            adj[pos].push_back(pos1);
          }
          if(j==1){
            if(pos1 < 0)
              pos1 = abs(pos1)+letterali;
            if(pos > letterali)
              pos = -(pos - letterali);
            adj[pos1].push_back(pos);
          }
          j++;
        }
      }
   // }

    for(int i = 1; i < nTotLet; i++){
      adj[i].sort();
      adj[i].unique();
    }
    list <int> :: iterator it1;
    for(int i = 1; i < nTotLet; i++){
      list <int> :: iterator it2;
      int val1 = 0;
      for(it1 = adj[i].begin(); it1 != adj[i].end(); it1++){
        val1 = *it1;
        if(val1 < 0){
          val1 = abs(val1) + letterali;
        }       
        for(it2 = adj[i].begin(); it2 != adj[i].end(); it2++){
          if(*it1!=*it2){
            adj2[val1].push_back(*it2);
            cout<<"Inserito "<<*it2<<" nella posizione "<<val1<<endl;
          }
        }
      }
    }

    

    for(int i = 1; i < nTotLet; i++){
      adj[i].merge(adj2[i]);
      adj[i].sort();
      adj[i].unique();
    }





    
    print(adj, nTotLet);
    
    
    /*int *d_matrice;
    cudaMalloc(&d_matrice, (vincoli*3)*sizeof(int));
    cudaMemcpy(d_matrice, matrice, (vincoli*3)*sizeof(int), cudaMemcpyHostToDevice);
    bool *CHECK, *d_CHECK;
    CHECK = (bool*)malloc(vincoli*sizeof(bool));
    
    cudaMalloc(&d_letter, letterali*sizeof(int)); 
    cudaMalloc(&d_CHECK, (vincoli)*sizeof(bool));
    
    cudaMemcpy(d_letter, letter, letterali*sizeof(int), cudaMemcpyHostToDevice);
    cudaMemcpy(d_CHECK, CHECK, (vincoli)*sizeof(bool), cudaMemcpyHostToDevice);
    
    bool *d_out;
    bool out;
    cudaMalloc(&d_out, sizeof(bool));
    
    int k = 0;
    i = 0;
    double elevamento = pow(2, letterali);
    //cout<<elevamento<<endl; 
    while(k<5 && valore<elevamento)
    { 
    cout<<"Sto facendo il valore "<<valore<<endl;
    cout<<valore/elevamento<<"%"<<endl;
    trasformaDecBin<<<40, 1024>>>(d_letter, valore, d_matrice, vincoli, d_CHECK, letterali);
    cudaDeviceSynchronize();
    accumulateKernel<<<40, 1024>>>(d_CHECK, vincoli, d_out);
    cudaDeviceSynchronize();
    cudaMemcpy(&out, d_out, sizeof(bool), cudaMemcpyDeviceToHost);
    if(out){  
    cout<<endl;
    cout<<"Ok per il valore "<<valore<<endl;
    k++;
  }
  valore++;
}
cout<<endl<<"Numero sol trovate: "<<k<<endl;

cudaFree(d_letter);
cudaFree(d_matrice);
cudaFree(d_out);
free(letter);

*/

return 0;
}




