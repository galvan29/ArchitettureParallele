#include "hip/hip_runtime.h"
/*

ArrayList<Integer[]> values = new ArrayList<>(); 
values.add(new Integer[] { 2, -4 }); 
int thid = blockIdx.x * blockDim.x + threadIdx.x;
int thidI = 0;
for(int Pass=0; Pass<ceilf((len/(blockDim.x)))+1; Pass++){
thidI = thid + Pass*(gridDim.x*blockDim.x );
if(thidI<len){

if(!CHECK[thidI] && *out){
*out=false;
}
}
}
*/

#include <stdio.h>
#include <iostream>
#include <cmath>
#include <fstream>
#include <bits/stdc++.h>
#include <numeric> 
#include <hip/hip_runtime.h>
#include <omp.h>
#include <unistd.h>
using namespace std;

__global__ void prova(int *d_adjMatrix, int len, int size, int *out){
  int thid = blockIdx.x * blockDim.x + threadIdx.x;
  /* if(thid==0){
    printf("%p %p",d_adjMatrix, d_adjMatrix[0]);
  } 
  if(thid < len && thid != 0)
  {
    int i = 0;
    printf("Ciao sono %d \n", thid);
    printf("Ciao prima mia posizione %d \n", d_adjMatrix[thid*size]);
    while(d_adjMatrix[(thid*size) +i ] != 0 && i<size){
      printf("Sono %d e sto guardando in pos %d il valore %d \n", thid, i, d_adjMatrix[(thid*size) +i]);
      i++;
    }
  }
  __syncthreads();
  if(thid == 0)
    for(int i = 0; i< len*size; i++){
      printf("%d in posizione %d\n",d_adjMatrix[i], i);
    }
  */

  // 1 TRUE  ---   -1 FALSE   --- 0 neutro
  out[1] = 1;
  if(thid == 0)
  {
      printf("Valore %d \n", out[1]);
    printf("Valore %d \n", out[12]);
  }

  __syncthreads();
}



void print(list<int>& mylist, int index)
{
  cout << "The list elements stored at the index " << 
  index << ": \n";
  
  for (auto element : mylist) 
  {
    cout<<element<<" ";
  }
  cout << '\n';
}

void print(list<int>* myContainer, int n)
{
  cout << "adj elements:\n\n";
  for (int i = 1; i < n; i++) 
  {
    print(myContainer[i], i);
  }
}

void funcRead(string str[]){
  string myText;
  int p = 0;
  ifstream MyReadFile("vincoli.txt");
  while (getline (MyReadFile, myText)) {
    str[p] = myText;
    p++;
  }
  MyReadFile.close();
}

string firstLine(){
  ifstream infile("vincoli.txt");
  string sLine;
  if (infile.good()){
    getline(infile, sLine);
  }
  return sLine;
}

int main(void)
{ 
  string s = firstLine();
  string arrayyy[4];
  stringstream ss(s);
  string word;
  int i=0;
  while (ss >> word) {
    arrayyy[i]=word;
    i++;
  }
  int letterali = stoi(arrayyy[2]);
  int vincoli = stoi(arrayyy[3]);
  int nTotLet = (letterali*2)+1;
  list<int> adj[nTotLet];
  list<int> adj2[nTotLet];
  
  string str[vincoli+1];
  funcRead(str);
  
  //int *letter, *d_letter;
  //letter = (int*)malloc(letterali*sizeof(int));
  
  // #pragma omp parallel shared(str, adj)
  // { 
  //  #pragma omp for schedule(auto)
  for(int i=1; i<(vincoli+1); i++){
    stringstream ss(str[i]);
    string word;
    int j = 0;
    int pos = 0;
    int pos1 = 0;
    
    while (ss >> word && j<=1) {
      if(j==0)
      pos = stoi(word);
      if(j==1){
        pos1=stoi(word);
        if(pos < 0)
        pos = abs(pos)+letterali;
        adj[pos].push_back(pos1);
      }
      if(j==1){
        if(pos1 < 0)
        pos1 = abs(pos1)+letterali;
        if(pos > letterali)
        pos = -(pos - letterali);
        adj[pos1].push_back(pos);
      }
      j++;
    }
  }
  // }
  
  for(int i = 1; i < nTotLet; i++){
    adj[i].sort();
    adj[i].unique();
  }
  list <int> :: iterator it1;
  for(int i = 1; i < nTotLet; i++){
    list <int> :: iterator it2;
    int val1 = 0;
    for(it1 = adj[i].begin(); it1 != adj[i].end(); it1++){
      val1 = *it1;
      if(val1 < 0){
        val1 = abs(val1) + letterali;
      }       
      for(it2 = adj[i].begin(); it2 != adj[i].end(); it2++){
        if(*it1!=*it2){
          adj2[val1].push_back(*it2);
          //cout<<"Inserito "<<*it2<<" nella posizione "<<val1<<endl;
        }
      }
    }
  }
  
  
  int size = 0;
  for(int i = 1; i < nTotLet; i++){
    adj[i].merge(adj2[i]);
    adj[i].sort();
    adj[i].unique();
    if(adj[i].size() > size)
      size = adj[i].size();
  }
  //utilizzo adj2
  int adjMatrix[nTotLet*size] = {0};
/*
  for(int i=1; i<nTotLet; i++){
    for(int j=0; j<2; j++){
      cout<<adjMatrix[i][j]<<" ";
    }
    cout<<endl;
  }
  cout<<endl; */

  //https://docs.nvidia.com/cuda/cusparse/index.html#coo-format

  for(int i=1; i<nTotLet; i++){
    int j = 0;
    while(adj[i].size() > 0){
      adjMatrix[(i*size) + j] = adj[i].front();
      adj[i].pop_front();
      j++;
    }
  } 



  //out potrebbe essere di bool tanto è true o false
  int *d_adjMatrix;
  hipMalloc(&d_adjMatrix, nTotLet*size*sizeof(int));
  hipMemcpy(d_adjMatrix, adjMatrix, nTotLet*size*sizeof(int), hipMemcpyHostToDevice);
  //sleep(1);
  int out[nTotLet];
  int *d_out;
  hipMalloc(&d_out, nTotLet*sizeof(int));
  prova<<<2, 32>>>(d_adjMatrix, nTotLet, size, d_out);
  hipMemcpy(&out, d_out, sizeof(int), hipMemcpyDeviceToHost);
  
  
  
  
  //print(adj, nTotLet);
  
  
  /*int *d_matrice;
  hipMalloc(&d_matrice, (vincoli*3)*sizeof(int));
  hipMemcpy(d_matrice, matrice, (vincoli*3)*sizeof(int), hipMemcpyHostToDevice);
  bool *CHECK, *d_CHECK;
  CHECK = (bool*)malloc(vincoli*sizeof(bool));
  
  hipMalloc(&d_letter, letterali*sizeof(int)); 
  hipMalloc(&d_CHECK, (vincoli)*sizeof(bool));
  
  hipMemcpy(d_letter, letter, letterali*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_CHECK, CHECK, (vincoli)*sizeof(bool), hipMemcpyHostToDevice);
  
  bool *d_out;
  bool out;
  hipMalloc(&d_out, sizeof(bool));
  
  int k = 0;
  i = 0;
  double elevamento = pow(2, letterali);
  //cout<<elevamento<<endl; 
  while(k<5 && valore<elevamento)
  { 
  cout<<"Sto facendo il valore "<<valore<<endl;
  cout<<valore/elevamento<<"%"<<endl;
  trasformaDecBin<<<40, 1024>>>(d_letter, valore, d_matrice, vincoli, d_CHECK, letterali);
  hipDeviceSynchronize();
  accumulateKernel<<<40, 1024>>>(d_CHECK, vincoli, d_out);
  hipDeviceSynchronize();
  hipMemcpy(&out, d_out, sizeof(bool), hipMemcpyDeviceToHost);
  if(out){  
  cout<<endl;
  cout<<"Ok per il valore "<<valore<<endl;
  k++;
}
valore++;
}
cout<<endl<<"Numero sol trovate: "<<k<<endl;

hipFree(d_letter);
hipFree(d_matrice);
hipFree(d_out);
free(letter);

*/

return 0;
}




