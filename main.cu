#include "2SAT_2.2.cu"
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <fstream>
#include <bits/stdc++.h>
#include <numeric>
#include <hip/hip_runtime.h>
#include <omp.h>
#include <malloc.h>
#include <unistd.h>
#include <list>


int main(void)  //main
{
  string s = firstLine();   //read first line of constraint file
  string infoFirstLine[4];
  stringstream ss(s);
  string word;
  int i = 0;
  while (ss >> word)     //save information
  {
    infoFirstLine[i] = word;
    i++;
  }

  int nLitt = stoi(infoFirstLine[2]);             //number of literals
  int nConstr = stoi(infoFirstLine[3]);           //number of constraints
  int nNegPosLit = (nLitt * 2);                  //number of literals (negative and positive)
  long int sizeAdj = nNegPosLit * nNegPosLit;     //size of adj matrix
  bool adj_matrix[sizeAdj] = {0};             //adj_matrix of all 0
  string str[nConstr + 1];
  funcRead(str);
  
  bool littExist[nNegPosLit] = {false};   //array of presence in the constraints
  // #pragma omp parallel shared(str, adj)
  // {
  //  #pragma omp for schedule(auto)

  for (int i = 1; i <= nConstr; i++)              //marks the existence of a litteral inside the constriants
  {
    stringstream ss(str[i]);
    string word;
    int pos = 0;
    int pos1 = 0;
    int j = 0;
    while (ss >> word && j <= 1)
    {
      if (j == 0)
      {
        pos = tras(stoi(word), nLitt);
      }
      else if (j == 1)
      {
        pos1 = tras(stoi(word), nLitt);
        adj_matrix[((pos * nNegPosLit) + pos1)] = 1;    //save the existence of constraints between two litterals 
        adj_matrix[((pos1 * nNegPosLit) + pos)] = 1;
      }
      littExist[pos] = true;
      littExist[pos1] = true;
      j++;
    }
  }
  //}


  bool *d_littExist;
  hipMalloc(&d_littExist, nNegPosLit * sizeof(bool));
  hipMemcpy(d_littExist, littExist, nNegPosLit * sizeof(bool), hipMemcpyHostToDevice);

  bool *d_adj_matrix;
  hipMalloc(&d_adj_matrix, sizeAdj * sizeof(bool));
  hipMemcpy(d_adj_matrix, adj_matrix, sizeAdj * sizeof(bool), hipMemcpyHostToDevice);

  hipDeviceSynchronize();

  int posizione[3] = {0};
  int *d_posizione;
  hipMalloc(&d_posizione, 3 * sizeof(int));

  //creo nuovi archi
  prova<<<40, 1024>>>(d_adj_matrix, nNegPosLit, sizeAdj, d_posizione);
  hipDeviceSynchronize();
  hipMemcpy(posizione, d_posizione, 3 * sizeof(int), hipMemcpyDeviceToHost);
  cout<<"Bro "<<posizione[0]<<endl;
  posizione[0] = 0;
  hipMemcpy(d_posizione, posizione, 3 * sizeof(int), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  prova<<<40, 1024>>>(d_adj_matrix, nNegPosLit, sizeAdj, d_posizione);
  hipDeviceSynchronize();
  hipMemcpy(posizione, d_posizione, 3 * sizeof(int), hipMemcpyDeviceToHost);
  cout<<"Bro "<<posizione[0]<<endl;
  hipDeviceSynchronize();
  //check modifiche
  posizione[0] = 0;
  hipMemcpy(d_posizione, posizione, 3 * sizeof(int), hipMemcpyHostToDevice);
          
  checkDiagonale<<<40, 1024>>>(d_adj_matrix, nNegPosLit);

  hipDeviceSynchronize();

  int sol[nNegPosLit] = {0};
  int *d_sol;
  hipMalloc(&d_sol, nNegPosLit * sizeof(int));
  int sol_backup[nNegPosLit] = {0};
  int *d_sol_backup;
  hipMalloc(&d_sol_backup, nNegPosLit * sizeof(int));

  int k = 3;
  int indexSol = 0;
  int cSol = 0;
  list<double> prox[1];
  int solReg[nNegPosLit * 1000];
  int *d_solReg;
  hipMalloc(&d_solReg, (nNegPosLit * 1000) * sizeof(int));
  hipMemcpy(d_solReg, solReg, (nNegPosLit * 1000) * sizeof(int), hipMemcpyHostToDevice);

  //array per soluzioni finali
  int solFinali[nNegPosLit * k];
  int *d_solFinali;
  hipMalloc(&d_solFinali, (nNegPosLit * k) * sizeof(int));
  hipMemcpy(d_solFinali, solFinali, (nNegPosLit * k) * sizeof(int), hipMemcpyHostToDevice);

  bool visitato[nNegPosLit] = {0};
  bool *d_visitato;

  hipMalloc(&d_visitato, nNegPosLit * sizeof(bool));
  hipMemcpy(d_visitato, visitato, nNegPosLit * sizeof(bool), hipMemcpyHostToDevice);
  hipMemcpy(d_sol, sol, nNegPosLit * sizeof(int), hipMemcpyHostToDevice);
  bool riprendoSoluzione = false;

  i = 0;

  bool esiste = false;
  bool continua = false;
  do{
    continua = false;
    hipMemcpy(d_sol, sol, nNegPosLit * sizeof(int), hipMemcpyHostToDevice);
    do{
      posizione[0] = 0;
      hipMemcpy(d_posizione, posizione, 3 * sizeof(int), hipMemcpyHostToDevice);
      if(sol[i] == 0 && sol[i+nLitt] == 0 && riprendoSoluzione == false){
        memcpy(sol_backup, sol, nNegPosLit*sizeof(int));
        if(littExist[i]){
          sol[i] = -1;
          sol_backup[i] = 1;
          esiste = true;
        }
        if(littExist[i + nLitt]){
          sol[i + nLitt] = 1;
          sol_backup[i + nLitt] = -1;
          esiste = true;
        }
        if(esiste){
          prox[0].push_back(i);
          hipMemcpy(d_sol_backup, sol_backup, nNegPosLit * sizeof(int), hipMemcpyHostToDevice);
          //printInt(sol_backup, nNegPosLit);
          salvaSoluzioneProx<<<40, 1024>>>(d_solReg, d_sol_backup, nNegPosLit, cSol);
          //cout<<cSol<<endl; 
          hipDeviceSynchronize();
          cSol++;
          hipMemcpy(d_sol, sol, nNegPosLit * sizeof(int), hipMemcpyHostToDevice);
        }
      }
      if(!riprendoSoluzione){
        if(esiste){
          riprendoSoluzione = true;   
          esiste = false;       
        }
        i++;
        cout<<i<<endl;
      }
      if(riprendoSoluzione){
        checkRow<<<40, 1024>>>(d_adj_matrix, d_sol, nNegPosLit, d_posizione, d_visitato, d_littExist);
        hipDeviceSynchronize();
        hipMemcpy(posizione, d_posizione, 3 * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(sol, d_sol, nNegPosLit * sizeof(int), hipMemcpyDeviceToHost);  //Da migliorare
        if(posizione[0] == 0)
          riprendoSoluzione = false;
      }
    }while(i < nLitt && posizione[1] == 0); 

    hipDeviceSynchronize();
    hipMemcpy(posizione, d_posizione, 3 * sizeof(int), hipMemcpyDeviceToHost);
    if(posizione[1] == 0){
      //cout<<"Trovata una soluzione"<<endl;
      //printInt(sol, nNegPosLit);
      completaSol<<<40, 1024>>>(d_sol, nNegPosLit, d_littExist);
      hipDeviceSynchronize();
      hipMemcpy(sol, d_sol, nNegPosLit * sizeof(int), hipMemcpyDeviceToHost); 
      /*cout<<"Soluzione sistemata"<<endl;
      for (int ssif = 0; ssif < nNegPosLit; ssif++)
      {   
          if(!littExist[ssif])
            cout<<-2<<" ";
          else
            cout << sol[ssif] << " ";
      }
      cout<<endl;
      cout<<endl;*/
      //SALVARE SE NUOVA
      //chiamo funzione 
      if(indexSol > 0)
        controlloNuovaSol<<<40, 1024>>>(d_sol, d_solFinali, nNegPosLit, indexSol, d_posizione);
      //cout<<"Valore indexaSol"<<endl;

      hipMemcpy(posizione, d_posizione, 3 * sizeof(int), hipMemcpyDeviceToHost);
      if(posizione[2] == 0 || indexSol == 0){
        k--;
        for (int ssif = 0; ssif < nNegPosLit; ssif++)
        {   
            solFinali[indexSol * nNegPosLit + ssif] = sol[ssif];
        }
        //printInt(sol, nNegPosLit);
        indexSol++;
        hipMemcpy(d_solFinali, solFinali, (nNegPosLit * k) * sizeof(int), hipMemcpyHostToDevice);
      }
     //cout<<"Array delle soluzioni"<<endl;
      //printInt(solFinali, nNegPosLit*(indexSol));
      posizione[2] = 0;
      hipMemcpy(d_posizione, posizione, 3 * sizeof(int), hipMemcpyHostToDevice);
    }

    posizione[1] = 0;
    hipMemcpy(d_posizione, posizione, 3 * sizeof(int), hipMemcpyHostToDevice);
    if (cSol > 0){
      memset(sol, 0, nNegPosLit * sizeof(int));
      i = prox[0].back();
      prox[0].pop_back();
      cSol--;
      copiaSoluzioneProx<<<40, 1024>>>(d_solReg, d_sol, nNegPosLit, cSol); 
      hipDeviceSynchronize();
      hipMemcpy(sol, d_sol, nNegPosLit * sizeof(int), hipMemcpyDeviceToHost);
      //printInt(sol, nNegPosLit);
      hipMemcpy(d_visitato, visitato, nNegPosLit * sizeof(bool), hipMemcpyHostToDevice);
      riprendoSoluzione = true;
      continua = true;
    }

  }while (continua && k > 0);

  ofstream myfile;
  myfile.open ("solution.txt");
  for(int ind = 0; ind < nNegPosLit*indexSol; ind++){
    myfile << solFinali[ind]<<" ";
    if(ind%nNegPosLit == (nNegPosLit-1) && ind != 0 && ind != (nNegPosLit*indexSol-1))
      myfile << "\n";
  }

  myfile.close();
  cout<<endl;
  cout<<"TERMINATO"<<endl;
  cout<<"k vale ora: "<<k<<endl;
  if(k == 0)
    cout<<"Ci sono tutte le soluzioni che cercavi"<<endl;
  hipFree(d_adj_matrix);
  hipFree(d_littExist);
  hipFree(d_posizione);
  hipFree(d_sol);
  hipFree(d_sol_backup);
  hipFree(d_solFinali);
  hipFree(d_solReg);
  hipFree(d_visitato);
  return 0;
}