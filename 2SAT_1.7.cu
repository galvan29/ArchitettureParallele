#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <fstream>
#include <bits/stdc++.h>
#include <numeric>
#include <hip/hip_runtime.h>
#include <omp.h>
#include <malloc.h>
#include <unistd.h>
#include <list>

using namespace std;

void funcRead(string str[])
{
  string myText;
  int p = 0;
  ifstream MyReadFile("vincoli.txt");
  while (getline(MyReadFile, myText))
  {
    str[p] = myText;
    p++;
  }
  MyReadFile.close();
}

string firstLine()
{
  ifstream infile("vincoli.txt");
  string sLine;
  if (infile.good())
  {
    getline(infile, sLine);
  }
  return sLine;
}

int tras(int number, int let)
{
  if (number < 0)
  {
    number = abs(number) + let;
  }
  return number - 1;
}

__device__ void sistema(bool *d_matrix, bool *d_matrix2, int length, int thid)
{
  if (d_matrix2[thid])
  {
    d_matrix[thid] = 1;
  }
}

__device__ void sistema2(bool *d_matrix, bool *d_matrix2, int length, int thid)
{
  d_matrix[thid] = d_matrix2[thid];
}

__device__ void diagonale(bool *d_matrix, int length, int thid)
{
  int secondo = (thid % length);
  int primo = floorf(thid / length);
  d_matrix[(primo * length) + secondo] |= d_matrix[(secondo * length) + primo];
}

__global__ void prova(bool *d_matrix, int length, long int lengthx2, bool *d_matrix2, bool *d_matrix3)
{
  int thid2 = blockIdx.x * blockDim.x + threadIdx.x;
  int thid = 0;

  for (int Pass = 0; Pass < ceilf((lengthx2 / (blockDim.x * gridDim.x))) + 1; Pass++)
  {
    thid = thid2 + Pass * (gridDim.x * blockDim.x);

    if (thid < (lengthx2))
    {
      if (d_matrix[thid])
      { // thid = 4
        int secondo = (thid % length);
        if (secondo >= (length / 2))
        {
          secondo = secondo - (length / 2);
        }
        else
        {
          secondo = secondo + (length / 2);
        }                                  // 4
        int primo = floorf(thid / length); // 0
        for (int i = (secondo * length); i < ((secondo + 1) * length); i++)
        { // da 24
          if (d_matrix[i] && ((i % length) + 1) != (primo + 1))
          { // 24 però 24%6+1 == 1 quindi non entro
            int posizione = (primo * length) + (i % length);
            d_matrix2[posizione] = 1;
          }
        }
      }
      sistema(d_matrix3, d_matrix2, length, thid);
      sistema2(d_matrix, d_matrix2, length, thid);
    }
    __syncthreads();

    if (thid < (length * length))
    {
      diagonale(d_matrix3, length, thid);
    }
  }
  __syncthreads();
}

// LA DIAGONALE NON MI TORNA
__global__ void checkDiagonale(bool *matrix, int length)
{
  int thid = blockIdx.x * blockDim.x + threadIdx.x;
  int thidCheck1 = 0;
  int thidCheck2 = 0;
  if (thid % (length + 1) == 0 && thid < (length * length))
  {
    thidCheck1 = thid;
    thidCheck2 = (length + 1) * (length / 2) + thid;
    if (matrix[thidCheck1] == 1 && matrix[thidCheck1] == matrix[thidCheck2])
    {
      printf("Nella posizione %d e nella posizione %d hanno entrambi 1\n", thidCheck1, thidCheck2);
      printf("Quindi teoricamente non ci sono soluzioni\n\n");
    }
  }
  if (thid == 0)
    printf("Ho controllato la diagonale\n");
  __syncthreads();
}

__global__ void daVisitare(bool *matrix, bool *d_daVis, int length, int index, int *d_sol)
{ // length è nLet tutto per due
  int thid2 = blockIdx.x * blockDim.x + threadIdx.x;
  int thid = 0;

  for (int Pass = 0; Pass < ceilf((length / (blockDim.x * gridDim.x))) + 1; Pass++)
  {
    thid = thid2 + Pass * (gridDim.x * blockDim.x);

    if (thid < length)
    {
      // printf("Sono %d di %d\n",thid, thid2);
      if (matrix[thid + index * length])
      {
        d_daVis[thid] = 1;
        d_sol[thid] = 1;
      }
    }
  }
  __syncthreads();
}

bool checkBoolArray(bool *daVis, int length)
{
  int i = 0;
  while (i < length)
  {
    if (daVis[i])
      return true;
    i++;
  }
  return false;
}

__global__ void completaSol(int *d_sol, int length)
{
  int thid2 = blockIdx.x * blockDim.x + threadIdx.x;
  int thid = 0;

  for (int Pass = 0; Pass < ceilf(((length / 2) / (blockDim.x * gridDim.x))) + 1; Pass++)
  {
    thid = thid2 + Pass * (gridDim.x * blockDim.x);
    if (thid < (length / 2))
    {
      if (d_sol[thid] == 0 && d_sol[thid + (length / 2)] != 0)
      {
        if (d_sol[thid + (length / 2)] == 1)
        {
          d_sol[thid] = -1;
        }
        else if (d_sol[thid + (length / 2)] == -1)
        {
          d_sol[thid] = 1;
        }
      }
      if (d_sol[thid] != 0 && d_sol[thid + (length / 2)] == 0)
      {
        if (d_sol[thid] == 1)
        {
          d_sol[thid + (length / 2)] = -1;
        }
        else if (d_sol[thid] == -1)
        {
          d_sol[thid + (length / 2)] = 1;
        }
      }
    }
  }
  __syncthreads();
}

__global__ void workVisit(bool *matrix, int *d_sol, bool *d_daVis, int index, int length, int *d_posizione, int *d_sol_backup)
{
  int valore = d_sol[index];
  d_daVis[index] = false;
  int thid2 = blockIdx.x * blockDim.x + threadIdx.x;
  int thid = 0;
  for (int Pass = 0; Pass < ceilf((length / (blockDim.x * gridDim.x))) + 1; Pass++)
  {
    thid = thid2 + Pass * (gridDim.x * blockDim.x);
    // thid = thid + (index*length);
    // printf("Io sono %d\n",thid);
    if (thid < length && d_daVis[thid])
    {
      // printf("VALORE VIDEO INIZIO %d \n", d_posizione[0]);
      // printf("Io sono %d\n",thid); // qua c'è qualcosa
      if (d_sol[thid] == 0 && valore == -1)
      {
        // printf("Io sono %d e il valore dentro d_sol[thid]: %d\n",thid, d_sol[thid]);
        d_daVis[thid] = true;
        d_sol[thid] = 1;
        // printf("Io sono %d e il valore dentro d_sol[thid]: %d\n",thid, d_sol[thid]);
      }
      else if (d_sol[thid] == 0 && valore == 1)
      {
        // printf("Io sono %d e il valore dentro d_sol[thid]: %d\n",thid, d_sol[thid]);
        d_daVis[thid] = true;
        d_sol[thid] = 1; // capire cosa fare qua, dovrei diramare ? -1
        // printf("Io sono %d e il valore dentro d_sol[thid]: %d\n",thid, d_sol[thid]);
      }
      else if (d_sol[thid] == (-1 * valore))
      {
        d_daVis[thid] = true;
      }
      else if (d_sol[thid] == valore && valore == -1)
      {
        // printf("Impossibile ottenere una soluzione grazie a %d\n", thid2);
        d_posizione[0] = d_posizione[0] || 1;
        // printf("Valore nella scheda video %d \n", d_posizione[0]);
      }
    }
    __syncthreads();
  }
  __syncthreads();
}

double trasformaDaArrayAIntNeg(int *sol, int length)
{
  double somma = 0;
  for (int i = (length - 1); i >= 0; i--)
  {
    if (sol[i] == -1)
    {
      somma += pow(2, (length - 1) - i);
    }
  }
  return somma;
}

double trasformaDaArrayAIntPos(int *sol, int length)
{
  double somma = 0;
  for (int i = (length - 1); i >= 0; i--)
  {
    if (sol[i] == 1)
    {
      somma += pow(2, (length - 1) - i);
    }
  }
  return somma;
}

void trasformaDaArrayAArray(int *sol, int length, int *temp)
{
  for (int i = 0; i < length; i++)
  {
    if (sol[i] == 1)
      sol[i] = -1;
    if (temp[i] == 1)
      sol[i] = 1;
  }
}

void trasformaDaIntAArray(int *sol, int length, int val)
{
  for (int i = (length - 1); i >= 0; i--)
  {
    if (val > 0)
    {
      sol[i] = val % 2;
      val = val / 2;
    }
    else
    {
      sol[i] = -1;
    }
  }
}

bool checkIfSolZero(int *sol, int nTotLet)
{
  for (int i = 0; i < nTotLet; i++)
  {
    if (sol[i] == 0)
    {
      return true;
    }
  }
  return true;
}

int main(void)
{
  string s = firstLine();
  string arrayyy[4];
  stringstream ss(s);
  string word;
  int i = 0;
  while (ss >> word)
  {
    arrayyy[i] = word;
    i++;
  }

  int letterali = stoi(arrayyy[2]);
  int vincoli = stoi(arrayyy[3]);
  int nTotLet = (letterali * 2);
  long int nTotLetx2 = nTotLet * nTotLet;
  bool matrix[nTotLetx2] = {0};
  string str[vincoli + 1];
  funcRead(str);

  // #pragma omp parallel shared(str, adj)
  // {
  //  #pragma omp for schedule(auto)
  for (int i = 1; i <= vincoli; i++)
  {
    stringstream ss(str[i]);
    string word;
    int pos = 0;
    int pos1 = 0;
    int j = 0;
    while (ss >> word && j <= 1)
    {
      if (j == 0)
      {
        pos = tras(stoi(word), letterali);
      }
      else if (j == 1)
      {
        pos1 = tras(stoi(word), letterali);
        matrix[((pos * nTotLet) + pos1)] = 1;
        matrix[((pos1 * nTotLet) + pos)] = 1;
      }
      j++;
    }
  }

  /*for(int i=0; i<nTotLetx2; i++){
    cout<<matrix[i]<<" ";
    if(i%nTotLet == (nTotLet-1))
      cout<<endl;
  }
  cout<<endl;
  //} */

  // https://docs.nvidia.com/cuda/cusparse/index.html#coo-format

  bool *d_matrix;
  bool *d_matrix2;
  bool *d_matrix3;

  hipMalloc(&d_matrix, nTotLetx2 * sizeof(bool));
  hipMalloc(&d_matrix2, nTotLetx2 * sizeof(bool));
  hipMalloc(&d_matrix3, nTotLetx2 * sizeof(bool));

  hipMemcpy(d_matrix, matrix, nTotLetx2 * sizeof(bool), hipMemcpyHostToDevice);
  hipMemcpy(d_matrix3, matrix, nTotLetx2 * sizeof(bool), hipMemcpyHostToDevice);
  sleep(10);
  // bool out[nTotLet];
  // bool *d_out;
  // hipMalloc(&d_out, nTotLetx2*sizeof(bool));

  hipDeviceSynchronize();

  prova<<<40, 1024>>>(d_matrix, nTotLet, nTotLetx2, d_matrix2, d_matrix3);
  hipDeviceSynchronize();
  hipFree(d_matrix);
  hipFree(d_matrix2);
  checkDiagonale<<<40, 1024>>>(d_matrix3, nTotLet);

  hipDeviceSynchronize();

  // PROVIAMO A CERCARE UNA SOLUZIONE
  int sol[nTotLet] = {0};
  int *d_sol;
  hipMalloc(&d_sol, nTotLet * sizeof(int));
  hipMemcpy(d_sol, sol, nTotLet * sizeof(int), hipMemcpyHostToDevice);
  int sol_backup[nTotLet] = {0};
  int *d_sol_backup;
  hipMalloc(&d_sol_backup, nTotLet * sizeof(int));

  bool daVis[nTotLet];
  bool *d_daVis;
  hipMalloc(&d_daVis, nTotLet * sizeof(bool));
  // hipMemcpy(d_daVis, daVis, nTotLet*sizeof(int), hipMemcpyHostToDevice);
  int posizione[1] = {0};
  int *d_posizione;
  hipMalloc(&d_posizione, 1 * sizeof(int));
  hipMemcpy(d_posizione, posizione, 1 * sizeof(int), hipMemcpyHostToDevice);

  list<double> prox[100];
  list<double> soluzioniRegistrate[1];
  /*sol[0] = -1;
  sol[0+letterali] = 1;

  sol_backup[0] = 1;
  sol_backup[0+letterali] = -1;


  //prox.push_back(trasformaDaArrayAInt(sol_backup, nTotLet));
  //prox.push_back(trasformaDaArrayAInt(sol, nTotLet));
  //cout<<"Questa è una soluzione da vedere "<<prox[0].back()<<endl; */
  bool riprendoSoluzione = false;
  int temp[nTotLet];

  do
  {
    if (prox[0].size() > 0)
      prox[0].pop_back();

    // lavoro
    bool alreadyC = true;
    int i = 0;
    do
    {
      if (!alreadyC)
      {
        hipMemcpy(sol, d_sol, nTotLet * sizeof(int), hipMemcpyDeviceToHost);
        alreadyC = true;
      }
      if (!riprendoSoluzione)
      {
        //cout<<i<<endl;
        if (sol[i] == 0 && sol[i + letterali] == 0)
        {
          sol[i] = -1;
          sol[i + letterali] = 1;

          sol_backup[i] = 1;
          sol_backup[i + letterali] = -1;
          prox[0].push_back(trasformaDaArrayAIntNeg(sol_backup, nTotLet));
          prox[0].push_back(i);
          prox[0].push_back(trasformaDaArrayAIntPos(sol_backup, nTotLet));
          riprendoSoluzione = true;
        }
        i++;
      }

      if (riprendoSoluzione)
      {
        hipMemcpy(d_sol, sol, nTotLet * sizeof(int), hipMemcpyHostToDevice);
        daVisitare<<<40, 1024>>>(d_matrix3, d_daVis, nTotLet, i, d_sol);
        hipDeviceSynchronize();
        hipMemcpy(daVis, d_daVis, nTotLet * sizeof(bool), hipMemcpyDeviceToHost);

        completaSol<<<40, 1024>>>(d_sol, nTotLet);
        hipDeviceSynchronize();

        int ind = 0;
        while (ind < nTotLet && checkBoolArray(daVis, nTotLet))
        {
          if (daVis[ind])
          {
            workVisit<<<40, 1024>>>(d_matrix3, d_sol, d_daVis, i, nTotLet, d_posizione, d_sol_backup);
            hipDeviceSynchronize(); // posizione da cui sono partito e valore che possiede
            hipMemcpy(daVis, d_daVis, nTotLet * sizeof(bool), hipMemcpyDeviceToHost);
          }
          if (ind == nTotLet)
            ind = 0;
          ind++;
        }

        completaSol<<<40, 1024>>>(d_sol, nTotLet);
        hipDeviceSynchronize();
        alreadyC = false;
        riprendoSoluzione = false;
      }

    } while (checkIfSolZero(sol, nTotLet) && i < nTotLet);

    // di una singola soluzione

    // appena finisce il ciclo devo andare a lavorare con un'altra soluzione che avevo.
    hipMemcpy(sol, d_sol, nTotLet * sizeof(int), hipMemcpyDeviceToHost);

    cout << endl
         << "Possibile soluzione :" << endl;
    for (int ssif = 0; ssif < nTotLet; ssif++)
    {
      cout << sol[ssif] << " ";
    }
    cout << endl;
    hipMemcpy(posizione, d_posizione, 1 * sizeof(int), hipMemcpyDeviceToHost);
    cout << "Valore delle discrepanze " << posizione[0] << endl;
    if (posizione[0] == 0)
    {
      soluzioniRegistrate[0].push_back(trasformaDaArrayAIntPos(sol, nTotLet));
    }
    posizione[0] = 0;
    hipMemcpy(d_posizione, posizione, 1 * sizeof(int), hipMemcpyHostToDevice);
    // hipMemset(d_posizione, 0, 1*sizeof(int));
    cout << endl;
    if (prox[0].size() > 0)
    {
      trasformaDaIntAArray(temp, nTotLet, prox[0].back());
      prox[0].pop_back();
      i = prox[0].back();
      prox[0].pop_back();
      trasformaDaIntAArray(sol, nTotLet, prox[0].back());
      trasformaDaArrayAArray(sol, nTotLet, temp);
    }
    cout<<"Sto riprendendo una soluzione, con i="<<i<<" questa:"<<endl;
    for(int bella = 0; bella < nTotLet; bella++){
      cout<<sol[bella]<< " ";
    }
    cout<<endl;
    riprendoSoluzione = true;
  } while (prox[0].size() > 0);

  // hipMemcpy(sol, d_sol, nTotLet*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(&matrix, d_matrix3, nTotLetx2 * sizeof(bool), hipMemcpyDeviceToHost);

  // hipMemcpy(&out, d_out, nTotLet*sizeof(int), hipMemcpyDeviceToHost);

  /*cout<<endl;
  for(int i=0; i<nTotLetx2; i++){
    cout<<matrix[i]<<" ";
    if(i%nTotLet == (nTotLet-1))
      cout<<endl;
  }
  cout<<endl;
  */

  cout << endl;
  cout << "Soluzioni mostrate in ordine di registrazione in valore intero: " << endl;
  cout << "Dal decimale al bin rendo 1 gli 1 e i -1 0: " << endl;
  soluzioniRegistrate[0].sort();
  soluzioniRegistrate[0].unique();
  while (soluzioniRegistrate[0].size() > 0)
  {
    cout << soluzioniRegistrate[0].front() << endl;
    soluzioniRegistrate[0].pop_front();
  }
  cout << endl;
  // double soluzNumerica = trasformaDaArrayAInt(sol, nTotLet);
  // cout<<soluzNumerica<<endl;

  hipFree(d_matrix3);
  return 0;
}
