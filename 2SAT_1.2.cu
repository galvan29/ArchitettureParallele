#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <fstream>
#include <bits/stdc++.h>
#include <numeric> 
#include <hip/hip_runtime.h>
#include <omp.h>
#pragma comment(linker, "/STACK:200000000")
#pragma comment(linker, "/HEAP:200000000")
using namespace std;

void funcRead(string str[]){
  string myText;
  int p = 0;
  ifstream MyReadFile("vincoli.txt");
  while (getline (MyReadFile, myText)) {
    str[p] = myText;
    p++;
  }
  MyReadFile.close();
}

string firstLine(){
  ifstream infile("vincoli.txt");
  string sLine;
  if (infile.good()){
    getline(infile, sLine);
  }
  return sLine;
}

int tras(int number, int let){
  if(number < 0){
    number = abs(number)+let;
  }
  return number-1;
}

__device__ void sistema(bool *d_matrix, bool *d_matrix2, int length, int thid){
  if(d_matrix2[thid]){
    d_matrix[thid]=1;
  }
}

__device__ void sistema2(bool *d_matrix, bool *d_matrix2, int length, int thid){
  d_matrix[thid]=d_matrix2[thid];
}

__device__ void diagonale(bool *d_matrix, int length, int thid){
  int secondo = (thid%length); 
  int primo = floorf(thid/length);
  d_matrix[(primo*length)+secondo] |= d_matrix[(secondo*length)+primo];
}

__global__ void prova(bool *d_matrix, int length, bool *d_matrix2, bool *d_matrix3){
  int thid2 = blockIdx.x * blockDim.x + threadIdx.x;
//length numero letterali
  int thid = 0;
  int doppio = length*length;

  for(int Pass=0; Pass<ceilf((doppio/(blockDim.x*gridDim.x)))+1; Pass++){
    thid = thid2 + Pass*(gridDim.x*blockDim.x);

    if(thid < (doppio)){
      if(d_matrix[thid]){  //thid = 4
        int secondo = (thid%length);  // 4
        int primo = floorf(thid/length);  // 0
        for(int i = (secondo*length); i < ((secondo+1)*length); i++){   //da 24
          //printf("%d %d %d\n", d_matrix[i], (primo+1), i);  
          if(d_matrix[i] && ((i%length)+1) != (primo+1)){    //24 però 24%6+1 == 1 quindi non entro
            int posizione = (primo*length) + (i%length);
            d_matrix2[posizione] = 1;
          }
        }
        int terzo = 0;
        if(secondo > (length/2)){
          terzo = secondo - (length/2);
        }else{
          terzo = secondo + (length/2);
        }
        for(int i = (terzo*length); i < ((terzo+1)*length); i++){   //da 24
          //printf("%d %d %d\n", d_matrix[i], (primo+1), i);  
          if(d_matrix[i] && ((i%length)+1) != (primo+1)){    //24 però 24%6+1 == 1 quindi non entro
            int posizione = (primo*length) + (i%length);
            d_matrix2[posizione] = 1;
          }
        }
      }
      sistema(d_matrix3, d_matrix2, length, thid);
      sistema2(d_matrix, d_matrix2, length, thid);
    }
    __syncthreads();

    if(thid < (length*length)){
      diagonale(d_matrix3, length, thid);
    }
  }
  __syncthreads();
}
//LA DIAGONALE NON MI TORNA
__global__ void checkDiagonale(bool *matrix, int length){
  int thid2 = blockIdx.x * blockDim.x + threadIdx.x;
  int thidCheck1 = 0;
  int thidCheck2 = 0;
  if(thid2%(length+1) == 0 && thid2 < (length*length)){
    thidCheck1 = thid2;
    thidCheck2 = (length+1)*(length/2) + thid2;
    if(matrix[thidCheck1] == 1 && matrix[thidCheck1] == matrix[thidCheck2]){
      printf("Nella posizione %d e nella posizione %d hanno entrambi 1\n", thidCheck1, thidCheck2);
      printf("Quindi teoricamente non ci sono soluzioni\n\n");
    }
  }
  if(thid2 == 0)
   printf("Ho controllato la diagonale\n");
  __syncthreads();  
}

int main(void)
{ 
  string s = firstLine();
  string arrayyy[4];
  stringstream ss(s);
  string word;
  int i=0;
  while (ss >> word) {
    arrayyy[i]=word;
    i++;
  }

  int letterali = stoi(arrayyy[2]);
  int vincoli = stoi(arrayyy[3]);
  int nTotLet = (letterali*2);
  int nTotLetx2 = nTotLet*nTotLet;
  bool matrix[nTotLetx2] = {0};
  string str[vincoli+1];
  funcRead(str);
  
  // #pragma omp parallel shared(str, adj)
  // { 
  //  #pragma omp for schedule(auto)
  for(int i=1; i<=vincoli; i++){
    stringstream ss(str[i]);
    string word;
    int pos = 0;
    int pos1 = 0;
    int j = 0;
    while (ss >> word && j<=1) {
      if(j==0){
        pos = tras(stoi(word), letterali);
      }
      else if(j == 1){
        pos1 = tras(stoi(word), letterali);
        matrix[((pos*nTotLet)+pos1)] = 1;
        matrix[((pos1*nTotLet)+pos)] = 1;
      }
      j++;
    }
  }
  /*for(int i=0; i<nTotLetx2; i++){
    cout<<matrix[i]<<" ";
    if(i%nTotLet == (nTotLet-1))
      cout<<endl;
  }
  cout<<endl;
  //}*/
  
  //https://docs.nvidia.com/cuda/cusparse/index.html#coo-format
  
  bool *d_matrix;
  bool *d_matrix2;
  bool *d_matrix3;
  hipMalloc(&d_matrix, nTotLetx2*sizeof(bool));
  hipMalloc(&d_matrix2, nTotLetx2*sizeof(bool));
  hipMalloc(&d_matrix3, nTotLetx2*sizeof(bool));
  hipDeviceSynchronize();
  hipMemcpy(d_matrix, matrix, nTotLetx2*sizeof(bool), hipMemcpyHostToDevice);
  hipMemcpy(d_matrix3, matrix, nTotLetx2*sizeof(bool), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  //bool out[nTotLet];
  //bool *d_out;
  //hipMalloc(&d_out, nTotLetx2*sizeof(bool));
  prova<<<40, 1024>>>(d_matrix, nTotLet, d_matrix2, d_matrix3);
  hipDeviceSynchronize();
  checkDiagonale<<<40, 1024>>>(d_matrix3, nTotLet);
  hipDeviceSynchronize();
  hipMemcpy(&matrix, d_matrix3, nTotLetx2*sizeof(bool), hipMemcpyDeviceToHost);
  //hipMemcpy(&out, d_out, nTotLet*sizeof(int), hipMemcpyDeviceToHost);
  
  for(int i=0; i<nTotLetx2; i++){
    cout<<matrix[i]<<" ";
    if(i%nTotLet == (nTotLet-1))
      cout<<endl;
  }
  cout<<endl;
  //}

  return 0;
}




